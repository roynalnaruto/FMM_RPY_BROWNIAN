#include "hip/hip_runtime.h"
/* Parallel Kernel Independent Fast Multipole Method
   Copyright (C) 2010 George Biros, Harper Langston, Ilya Lashuk
   Copyright (C) 2010, Aparna Chandramowlishwaran, Aashay Shingrapure, Rich Vuduc

This program is free software; you can redistribute it and/or modify
it under the terms of the GNU General Public License as published by
the Free Software Foundation; either version 2, or (at your option)
any later version.

This program is distributed in the hope that it will be useful, but WITHOUT
ANY WARRANTY; without even the implied warranty of MERCHANTABILITY or
FITNESS FOR A PARTICULAR PURPOSE.  See the GNU General Public License
for more details.

You should have received a copy of the GNU General Public License
along with this program; see the file COPYING.  If not, write to the Free
Software Foundation, Inc., 59 Temple Place - Suite 330, Boston, MA
02111-1307, USA.  */

using namespace std;
#include <mpi.h>

#define PI_4I 0.079577471F
#define PI_8I 0.0397887358F

//#define PI3_4I 0.238732413F
#include <cutil.h>
//#include <cutil_inline.h>
#include "../p3d/upComp.h"
#include "../p3d/dnComp.h"
#include "gpu_setup.h"
#include "kernel3d_mpi.hpp"

#include <cstdio>
#define MPI_ASSERT(c)  mpi_assert__ (((long)c), #c, __FILE__, __LINE__)

static
void
mpi_assert__ (long cond, const char* str_cond, const char* file, size_t line)
{
  if (!cond) {
    int rank;
    char procname[MPI_MAX_PROCESSOR_NAME+1];
    int procnamelen;
    memset (procname, 0, sizeof (procname));
    MPI_Comm_rank (MPI_COMM_WORLD, &rank);
    MPI_Get_processor_name (procname, &procnamelen);
    fprintf (stderr, "*** [%s:%lu--p%d(%s)] ASSERTION FAILURE: %s ***\n",
	     file, (unsigned long)line, rank, procname, str_cond);
    fflush (stderr);
    MPI_Abort (MPI_COMM_WORLD, 1);
  }
}

#define BLOCK_HEIGHT 64

__constant__ float3 sampos[320];	//undefined for everything greater than 295 for 6, greater than 191 for 4

__constant__ float3 samposDn[152];	//undefined for everything greater than 151 for 6 and 55 for 4

__global__ void up_kernel(float *src_dp,float *trgVal_dp,float *trgCtr_dp,float *trgRad_dp,int *srcBox_dp,int numSrcBox) {
	__shared__ float4 s_sh[BLOCK_HEIGHT];

	int uniqueBlockId=blockIdx.y * gridDim.x + blockIdx.x;
	if(uniqueBlockId<numSrcBox) {
		float3 trgCtr;
		float trgRad;
	//	float3 samp[5];
		float3 trg[5];
		float dX_reg;
		float dY_reg;
		float dZ_reg;
		int2 src=((int2*)srcBox_dp)[uniqueBlockId];	//x has start, y has size
		src.x+=threadIdx.x;

		trgCtr=((float3*)trgCtr_dp)[uniqueBlockId];
		trgRad=trgRad_dp[uniqueBlockId];

		//construct the trg

		trg[0].x=trgCtr.x+trgRad*sampos[4*threadIdx.x].x;
		trg[0].y=trgCtr.y+trgRad*sampos[4*threadIdx.x].y;
		trg[0].z=trgCtr.z+trgRad*sampos[4*threadIdx.x].z;
		trg[1].x=trgCtr.x+trgRad*sampos[4*threadIdx.x+1].x;
		trg[1].y=trgCtr.y+trgRad*sampos[4*threadIdx.x+1].y;
		trg[1].z=trgCtr.z+trgRad*sampos[4*threadIdx.x+1].z;
		trg[2].x=trgCtr.x+trgRad*sampos[4*threadIdx.x+2].x;
		trg[2].y=trgCtr.y+trgRad*sampos[4*threadIdx.x+2].y;
		trg[2].z=trgCtr.z+trgRad*sampos[4*threadIdx.x+2].z;
		trg[3].x=trgCtr.x+trgRad*sampos[4*threadIdx.x+3].x;
		trg[3].y=trgCtr.y+trgRad*sampos[4*threadIdx.x+3].y;
		trg[3].z=trgCtr.z+trgRad*sampos[4*threadIdx.x+3].z;
		trg[4].x=trgCtr.x+trgRad*sampos[256+threadIdx.x].x;
		trg[4].y=trgCtr.y+trgRad*sampos[256+threadIdx.x].y;
		trg[4].z=trgCtr.z+trgRad*sampos[256+threadIdx.x].z;

	//	int numSrc=srcBoxSize[uniqueBlockId];

		float4 tv=make_float4(0.0F,0.0F,0.0F,0.0F);
		float tve=0.0F;






		int num_chunk_loop=src.y/BLOCK_HEIGHT;
		for(int chunk=0;chunk<num_chunk_loop;chunk++) {
			__syncthreads();
			s_sh[threadIdx.x]=((float4*)src_dp)[src.x];
			__syncthreads();

			src.x+=BLOCK_HEIGHT;

			for(int s=0;s<BLOCK_HEIGHT;s++) {
				dX_reg=s_sh[s].x-trg[0].x;
				dY_reg=s_sh[s].y-trg[0].y;
				dZ_reg=s_sh[s].z-trg[0].z;

				dX_reg*=dX_reg;
				dY_reg*=dY_reg;
				dZ_reg*=dZ_reg;

				dX_reg += dY_reg+dZ_reg;

				dX_reg = rsqrtf(dX_reg);
//@@
dX_reg = dX_reg + (dX_reg-dX_reg);
dX_reg = fmaxf(dX_reg,0.0F);
				tv.x+=dX_reg*s_sh[s].w;
				///////////////////////////////
				dX_reg=s_sh[s].x-trg[1].x;
				dY_reg=s_sh[s].y-trg[1].y;
				dZ_reg=s_sh[s].z-trg[1].z;

				dX_reg*=dX_reg;
				dY_reg*=dY_reg;
				dZ_reg*=dZ_reg;

				dX_reg += dY_reg+dZ_reg;

				dX_reg = rsqrtf(dX_reg);
//@@
dX_reg = dX_reg + (dX_reg-dX_reg);
dX_reg = fmaxf(dX_reg,0.0F);
				tv.y+=dX_reg*s_sh[s].w;
				///////////////////////////////
				dX_reg=s_sh[s].x-trg[2].x;
				dY_reg=s_sh[s].y-trg[2].y;
				dZ_reg=s_sh[s].z-trg[2].z;

				dX_reg*=dX_reg;
				dY_reg*=dY_reg;
				dZ_reg*=dZ_reg;

				dX_reg += dY_reg+dZ_reg;

				dX_reg = rsqrtf(dX_reg);
//@@
dX_reg = dX_reg + (dX_reg-dX_reg);
dX_reg = fmaxf(dX_reg,0.0F);
				tv.z+=dX_reg*s_sh[s].w;
				///////////////////////////////
				dX_reg=s_sh[s].x-trg[3].x;
				dY_reg=s_sh[s].y-trg[3].y;
				dZ_reg=s_sh[s].z-trg[3].z;

				dX_reg*=dX_reg;
				dY_reg*=dY_reg;
				dZ_reg*=dZ_reg;

				dX_reg += dY_reg+dZ_reg;

				dX_reg = rsqrtf(dX_reg);
//@@
dX_reg = dX_reg + (dX_reg-dX_reg);
dX_reg = fmaxf(dX_reg,0.0F);
				tv.w+=dX_reg*s_sh[s].w;
				///////////////////////////////
				dX_reg=s_sh[s].x-trg[4].x;
				dY_reg=s_sh[s].y-trg[4].y;
				dZ_reg=s_sh[s].z-trg[4].z;

				dX_reg*=dX_reg;
				dY_reg*=dY_reg;
				dZ_reg*=dZ_reg;

				dX_reg += dY_reg+dZ_reg;

				dX_reg = rsqrtf(dX_reg);
//@@
dX_reg = dX_reg + (dX_reg-dX_reg);
dX_reg = fmaxf(dX_reg,0.0F);
				tve+=dX_reg*s_sh[s].w;
				///////////////////////////////
			}

		}	//end num chunk loop
		__syncthreads();
		s_sh[threadIdx.x]=((float4*)src_dp)[src.x];
		__syncthreads();
		for(int s=0;s<src.y%BLOCK_HEIGHT;s++) {
			dX_reg=s_sh[s].x-trg[0].x;
			dY_reg=s_sh[s].y-trg[0].y;
			dZ_reg=s_sh[s].z-trg[0].z;

			dX_reg*=dX_reg;
			dY_reg*=dY_reg;
			dZ_reg*=dZ_reg;

			dX_reg += dY_reg+dZ_reg;

			dX_reg = rsqrtf(dX_reg);
//@@
dX_reg = dX_reg + (dX_reg-dX_reg);
dX_reg = fmaxf(dX_reg,0.0F);
			tv.x+=dX_reg*s_sh[s].w;
			///////////////////////////////
			dX_reg=s_sh[s].x-trg[1].x;
			dY_reg=s_sh[s].y-trg[1].y;
			dZ_reg=s_sh[s].z-trg[1].z;

			dX_reg*=dX_reg;
			dY_reg*=dY_reg;
			dZ_reg*=dZ_reg;

			dX_reg += dY_reg+dZ_reg;

			dX_reg = rsqrtf(dX_reg);
//@@
dX_reg = dX_reg + (dX_reg-dX_reg);
dX_reg = fmaxf(dX_reg,0.0F);
			tv.y+=dX_reg*s_sh[s].w;
			///////////////////////////////
			dX_reg=s_sh[s].x-trg[2].x;
			dY_reg=s_sh[s].y-trg[2].y;
			dZ_reg=s_sh[s].z-trg[2].z;

			dX_reg*=dX_reg;
			dY_reg*=dY_reg;
			dZ_reg*=dZ_reg;

			dX_reg += dY_reg+dZ_reg;

			dX_reg = rsqrtf(dX_reg);
//@@
dX_reg = dX_reg + (dX_reg-dX_reg);
dX_reg = fmaxf(dX_reg,0.0F);
			tv.z+=dX_reg*s_sh[s].w;
			///////////////////////////////
			dX_reg=s_sh[s].x-trg[3].x;
			dY_reg=s_sh[s].y-trg[3].y;
			dZ_reg=s_sh[s].z-trg[3].z;

			dX_reg*=dX_reg;
			dY_reg*=dY_reg;
			dZ_reg*=dZ_reg;

			dX_reg += dY_reg+dZ_reg;

			dX_reg = rsqrtf(dX_reg);
//@@
dX_reg = dX_reg + (dX_reg-dX_reg);
dX_reg = fmaxf(dX_reg,0.0F);
			tv.w+=dX_reg*s_sh[s].w;
			///////////////////////////////
			dX_reg=s_sh[s].x-trg[4].x;
			dY_reg=s_sh[s].y-trg[4].y;
			dZ_reg=s_sh[s].z-trg[4].z;

			dX_reg*=dX_reg;
			dY_reg*=dY_reg;
			dZ_reg*=dZ_reg;

			dX_reg += dY_reg+dZ_reg;

			dX_reg = rsqrtf(dX_reg);
//@@
dX_reg = dX_reg + (dX_reg-dX_reg);
dX_reg = fmaxf(dX_reg,0.0F);
			tve+=dX_reg*s_sh[s].w;
			///////////////////////////////
		}	//end residual loop

		//write back
		tv.x*=PI_4I;
		tv.y*=PI_4I;
		tv.z*=PI_4I;
		tv.w*=PI_4I;
	//	tv.x=(float)trgCtr;
	//	tv.y=tv.z=tv.w=0.0F;
		((float4*)trgVal_dp)[uniqueBlockId*74+threadIdx.x]=tv;
		if(threadIdx.x<40)
			trgVal_dp[uniqueBlockId*296+256+threadIdx.x]=tve*PI_4I;
	}

}

__global__ void up_kernel_4(float *src_dp,float *trgVal_dp,float *trgCtr_dp,float *trgRad_dp,int *srcBox_dp,int numSrcBox) {
	__shared__ float4 s_sh[BLOCK_HEIGHT];

	int uniqueBlockId=blockIdx.y * gridDim.x + blockIdx.x;
	if(uniqueBlockId<numSrcBox) {
		float3 trgCtr;
		float trgRad;
	//	float3 samp[5];
		float3 trg[3];
		float dX_reg;
		float dY_reg;
		float dZ_reg;
		int2 src=((int2*)srcBox_dp)[uniqueBlockId];	//x has start, y has size
		src.x+=threadIdx.x;

		trgCtr=((float3*)trgCtr_dp)[uniqueBlockId];
		trgRad=trgRad_dp[uniqueBlockId];

		//construct the trg

		trg[0].x=trgCtr.x+trgRad*sampos[2*threadIdx.x].x;
		trg[0].y=trgCtr.y+trgRad*sampos[2*threadIdx.x].y;
		trg[0].z=trgCtr.z+trgRad*sampos[2*threadIdx.x].z;
		trg[1].x=trgCtr.x+trgRad*sampos[2*threadIdx.x+1].x;
		trg[1].y=trgCtr.y+trgRad*sampos[2*threadIdx.x+1].y;
		trg[1].z=trgCtr.z+trgRad*sampos[2*threadIdx.x+1].z;
		trg[2].x=trgCtr.x+trgRad*sampos[128+threadIdx.x].x;		//128 is blockheight*(trg2fetch-1)
		trg[2].y=trgCtr.y+trgRad*sampos[128+threadIdx.x].y;
		trg[2].z=trgCtr.z+trgRad*sampos[128+threadIdx.x].z;

	//	int numSrc=srcBoxSize[uniqueBlockId];

		float2 tv=make_float2(0.0F,0.0F);					//can be converted into a generic array.. not too big
		float tve=0.0F;






		int num_chunk_loop=src.y/BLOCK_HEIGHT;
		for(int chunk=0;chunk<num_chunk_loop;chunk++) {
			__syncthreads();
			s_sh[threadIdx.x]=((float4*)src_dp)[src.x];
			__syncthreads();

			src.x+=BLOCK_HEIGHT;

			for(int s=0;s<BLOCK_HEIGHT;s++) {
				dX_reg=s_sh[s].x-trg[0].x;
				dY_reg=s_sh[s].y-trg[0].y;
				dZ_reg=s_sh[s].z-trg[0].z;

				dX_reg*=dX_reg;
				dY_reg*=dY_reg;
				dZ_reg*=dZ_reg;

				dX_reg += dY_reg+dZ_reg;

				dX_reg = rsqrtf(dX_reg);
//@@
dX_reg = dX_reg + (dX_reg-dX_reg);
dX_reg = fmaxf(dX_reg,0.0F);
				tv.x+=dX_reg*s_sh[s].w;
				///////////////////////////////
				dX_reg=s_sh[s].x-trg[1].x;
				dY_reg=s_sh[s].y-trg[1].y;
				dZ_reg=s_sh[s].z-trg[1].z;

				dX_reg*=dX_reg;
				dY_reg*=dY_reg;
				dZ_reg*=dZ_reg;

				dX_reg += dY_reg+dZ_reg;

				dX_reg = rsqrtf(dX_reg);
//@@
dX_reg = dX_reg + (dX_reg-dX_reg);
dX_reg = fmaxf(dX_reg,0.0F);
				tv.y+=dX_reg*s_sh[s].w;
				///////////////////////////////
				dX_reg=s_sh[s].x-trg[2].x;
				dY_reg=s_sh[s].y-trg[2].y;
				dZ_reg=s_sh[s].z-trg[2].z;

				dX_reg*=dX_reg;
				dY_reg*=dY_reg;
				dZ_reg*=dZ_reg;

				dX_reg += dY_reg+dZ_reg;

				dX_reg = rsqrtf(dX_reg);
//@@
dX_reg = dX_reg + (dX_reg-dX_reg);
dX_reg = fmaxf(dX_reg,0.0F);
				tve+=dX_reg*s_sh[s].w;
			}
		}	//end num chunk loop
		__syncthreads();
		s_sh[threadIdx.x]=((float4*)src_dp)[src.x];
		__syncthreads();
		for(int s=0;s<src.y%BLOCK_HEIGHT;s++) {
			dX_reg=s_sh[s].x-trg[0].x;
			dY_reg=s_sh[s].y-trg[0].y;
			dZ_reg=s_sh[s].z-trg[0].z;

			dX_reg*=dX_reg;
			dY_reg*=dY_reg;
			dZ_reg*=dZ_reg;

			dX_reg += dY_reg+dZ_reg;

			dX_reg = rsqrtf(dX_reg);
//@@
dX_reg = dX_reg + (dX_reg-dX_reg);
dX_reg = fmaxf(dX_reg,0.0F);
			tv.x+=dX_reg*s_sh[s].w;
			///////////////////////////////
			dX_reg=s_sh[s].x-trg[1].x;
			dY_reg=s_sh[s].y-trg[1].y;
			dZ_reg=s_sh[s].z-trg[1].z;

			dX_reg*=dX_reg;
			dY_reg*=dY_reg;
			dZ_reg*=dZ_reg;

			dX_reg += dY_reg+dZ_reg;

			dX_reg = rsqrtf(dX_reg);
//@@
dX_reg = dX_reg + (dX_reg-dX_reg);
dX_reg = fmaxf(dX_reg,0.0F);
			tv.y+=dX_reg*s_sh[s].w;
			///////////////////////////////
			dX_reg=s_sh[s].x-trg[2].x;
			dY_reg=s_sh[s].y-trg[2].y;
			dZ_reg=s_sh[s].z-trg[2].z;

			dX_reg*=dX_reg;
			dY_reg*=dY_reg;
			dZ_reg*=dZ_reg;

			dX_reg += dY_reg+dZ_reg;

			dX_reg = rsqrtf(dX_reg);
//@@
dX_reg = dX_reg + (dX_reg-dX_reg);
dX_reg = fmaxf(dX_reg,0.0F);
			tve+=dX_reg*s_sh[s].w;
			///////////////////////////////
		}	//end residual loop

		//write back
		tv.x*=PI_4I;
		tv.y*=PI_4I;
	//	tv.x=(float)trgCtr;
	//	tv.y=tv.z=tv.w=0.0F;
		((float2*)(trgVal_dp+uniqueBlockId*152))[threadIdx.x]=tv;	//in generic, float3 writes will be unrolled into multiple writes
		if(threadIdx.x<24)
			trgVal_dp[uniqueBlockId*152+128+threadIdx.x]=tve*PI_4I;
	}

}

__global__ void up_kernel_stokes_velocity_4(float *src_dp,float *trgVal_dp,float *trgCtr_dp,float *trgRad_dp,int *srcBox_dp,int numSrcBox)
{
  __shared__ float3 sc_sh[BLOCK_HEIGHT]; // source coordinates
  __shared__ float3 sd_sh[BLOCK_HEIGHT]; // source densities

  int uniqueBlockId=blockIdx.y * gridDim.x + blockIdx.x;
  if(uniqueBlockId<numSrcBox) {
    float3 trgCtr;
    float trgRad;
    //	float3 samp[5];
    float3 trg[3];
    float dX_reg;
    float dY_reg;
    float dZ_reg;
    int2 src=((int2*)srcBox_dp)[uniqueBlockId];	//x has start, y has size
    src.x+=threadIdx.x;

    trgCtr=((float3*)trgCtr_dp)[uniqueBlockId];
    trgRad=trgRad_dp[uniqueBlockId];

    //construct the trg

    trg[0].x=trgCtr.x+trgRad*sampos[2*threadIdx.x].x;
    trg[0].y=trgCtr.y+trgRad*sampos[2*threadIdx.x].y;
    trg[0].z=trgCtr.z+trgRad*sampos[2*threadIdx.x].z;
    trg[1].x=trgCtr.x+trgRad*sampos[2*threadIdx.x+1].x;
    trg[1].y=trgCtr.y+trgRad*sampos[2*threadIdx.x+1].y;
    trg[1].z=trgCtr.z+trgRad*sampos[2*threadIdx.x+1].z;
    trg[2].x=trgCtr.x+trgRad*sampos[128+threadIdx.x].x;		//128 is blockheight*(trg2fetch-1)
    trg[2].y=trgCtr.y+trgRad*sampos[128+threadIdx.x].y;
    trg[2].z=trgCtr.z+trgRad*sampos[128+threadIdx.x].z;

    //	int numSrc=srcBoxSize[uniqueBlockId];

    float3 pot0=make_float3(0.0F,0.0F,0.0F);
    float3 pot1=make_float3(0.0F,0.0F,0.0F);
    float3 pot2=make_float3(0.0F,0.0F,0.0F);

    int num_chunk_loop=src.y/BLOCK_HEIGHT;
    for(int chunk=0;chunk<num_chunk_loop;chunk++) {
      __syncthreads();
      sc_sh[threadIdx.x]=((float3*)src_dp)[2*src.x];
      sd_sh[threadIdx.x]=((float3*)src_dp)[2*src.x+1];
      __syncthreads();

      src.x+=BLOCK_HEIGHT;

      for(int s=0;s<BLOCK_HEIGHT;s++) {
	dX_reg=sc_sh[s].x-trg[0].x;
	dY_reg=sc_sh[s].y-trg[0].y;
	dZ_reg=sc_sh[s].z-trg[0].z;

	float invR = rsqrtf(dX_reg*dX_reg+dY_reg*dY_reg+dZ_reg*dZ_reg);

	// following two lines set invR to zero if invR is infinity
	invR = invR + (invR-invR);
	invR = fmaxf(invR, 0.0F);

	float3 cur_pot = sd_sh[s];
	float tmp_scalar = (dX_reg*cur_pot.x + dY_reg*cur_pot.y + dZ_reg*cur_pot.z)*invR*invR;
	cur_pot.x += tmp_scalar*dX_reg;
	cur_pot.y += tmp_scalar*dY_reg;
	cur_pot.z += tmp_scalar*dZ_reg;

	pot0.x += cur_pot.x*invR;
	pot0.y += cur_pot.y*invR;
	pot0.z += cur_pot.z*invR;

	///////////////////////////////
	
	dX_reg=sc_sh[s].x-trg[1].x;
	dY_reg=sc_sh[s].y-trg[1].y;
	dZ_reg=sc_sh[s].z-trg[1].z;

	invR = rsqrtf(dX_reg*dX_reg+dY_reg*dY_reg+dZ_reg*dZ_reg);

	// following two lines set invR to zero if invR is infinity
	invR = invR + (invR-invR);
	invR = fmaxf(invR, 0.0F);

	cur_pot = sd_sh[s];
	tmp_scalar = (dX_reg*cur_pot.x + dY_reg*cur_pot.y + dZ_reg*cur_pot.z)*invR*invR;
	cur_pot.x += tmp_scalar*dX_reg;
	cur_pot.y += tmp_scalar*dY_reg;
	cur_pot.z += tmp_scalar*dZ_reg;

	pot1.x += cur_pot.x*invR;
	pot1.y += cur_pot.y*invR;
	pot1.z += cur_pot.z*invR;

	///////////////////////////////
	
	dX_reg=sc_sh[s].x-trg[2].x;
	dY_reg=sc_sh[s].y-trg[2].y;
	dZ_reg=sc_sh[s].z-trg[2].z;

	invR = rsqrtf(dX_reg*dX_reg+dY_reg*dY_reg+dZ_reg*dZ_reg);

	// following two lines set invR to zero if invR is infinity
	invR = invR + (invR-invR);
	invR = fmaxf(invR, 0.0F);

	cur_pot = sd_sh[s];
	tmp_scalar = (dX_reg*cur_pot.x + dY_reg*cur_pot.y + dZ_reg*cur_pot.z)*invR*invR;
	cur_pot.x += tmp_scalar*dX_reg;
	cur_pot.y += tmp_scalar*dY_reg;
	cur_pot.z += tmp_scalar*dZ_reg;

	pot2.x += cur_pot.x*invR;
	pot2.y += cur_pot.y*invR;
	pot2.z += cur_pot.z*invR;
      }
    }	//end num chunk loop
    __syncthreads();
      sc_sh[threadIdx.x]=((float3*)src_dp)[2*src.x];
      sd_sh[threadIdx.x]=((float3*)src_dp)[2*src.x+1];
    __syncthreads();
    for(int s=0;s<src.y%BLOCK_HEIGHT;s++) {
	dX_reg=sc_sh[s].x-trg[0].x;
	dY_reg=sc_sh[s].y-trg[0].y;
	dZ_reg=sc_sh[s].z-trg[0].z;

	float invR = rsqrtf(dX_reg*dX_reg+dY_reg*dY_reg+dZ_reg*dZ_reg);

	// following two lines set invR to zero if invR is infinity
	invR = invR + (invR-invR);
	invR = fmaxf(invR, 0.0F);

	float3 cur_pot = sd_sh[s];
	float tmp_scalar = (dX_reg*cur_pot.x + dY_reg*cur_pot.y + dZ_reg*cur_pot.z)*invR*invR;
	cur_pot.x += tmp_scalar*dX_reg;
	cur_pot.y += tmp_scalar*dY_reg;
	cur_pot.z += tmp_scalar*dZ_reg;

	pot0.x += cur_pot.x*invR;
	pot0.y += cur_pot.y*invR;
	pot0.z += cur_pot.z*invR;

	///////////////////////////////
	
	dX_reg=sc_sh[s].x-trg[1].x;
	dY_reg=sc_sh[s].y-trg[1].y;
	dZ_reg=sc_sh[s].z-trg[1].z;

	invR = rsqrtf(dX_reg*dX_reg+dY_reg*dY_reg+dZ_reg*dZ_reg);

	// following two lines set invR to zero if invR is infinity
	invR = invR + (invR-invR);
	invR = fmaxf(invR, 0.0F);

	cur_pot = sd_sh[s];
	tmp_scalar = (dX_reg*cur_pot.x + dY_reg*cur_pot.y + dZ_reg*cur_pot.z)*invR*invR;
	cur_pot.x += tmp_scalar*dX_reg;
	cur_pot.y += tmp_scalar*dY_reg;
	cur_pot.z += tmp_scalar*dZ_reg;

	pot1.x += cur_pot.x*invR;
	pot1.y += cur_pot.y*invR;
	pot1.z += cur_pot.z*invR;

	///////////////////////////////
	
	dX_reg=sc_sh[s].x-trg[2].x;
	dY_reg=sc_sh[s].y-trg[2].y;
	dZ_reg=sc_sh[s].z-trg[2].z;

	invR = rsqrtf(dX_reg*dX_reg+dY_reg*dY_reg+dZ_reg*dZ_reg);

	// following two lines set invR to zero if invR is infinity
	invR = invR + (invR-invR);
	invR = fmaxf(invR, 0.0F);

	cur_pot = sd_sh[s];
	tmp_scalar = (dX_reg*cur_pot.x + dY_reg*cur_pot.y + dZ_reg*cur_pot.z)*invR*invR;
	cur_pot.x += tmp_scalar*dX_reg;
	cur_pot.y += tmp_scalar*dY_reg;
	cur_pot.z += tmp_scalar*dZ_reg;

	pot2.x += cur_pot.x*invR;
	pot2.y += cur_pot.y*invR;
	pot2.z += cur_pot.z*invR;
    }	//end residual loop

    //write back
    pot0.x *= PI_8I;
    pot0.y *= PI_8I;
    pot0.z *= PI_8I;
    pot1.x *= PI_8I;
    pot1.y *= PI_8I;
    pot1.z *= PI_8I;
    pot2.x *= PI_8I;
    pot2.y *= PI_8I;
    pot2.z *= PI_8I;

    ((float3*)trgVal_dp)[uniqueBlockId*152+2*threadIdx.x]=pot0;
    ((float3*)trgVal_dp)[uniqueBlockId*152+2*threadIdx.x+1]=pot1;

    if(threadIdx.x<24)
      ((float3*)trgVal_dp)[uniqueBlockId*152+128+threadIdx.x]=pot2;
  }
}

// void unmake_ds_up(float *trgValE,upComp_t *UpC) {
// 	int t=0;
// 	for(int i=0;i<UpC->numSrcBox;i++) {
// 		for(int j=0;j<UpC->trgDim;j++) {
// //			assert(UpC->trgVal[i]!=NULL);
// 
// 			if(UpC->trgVal[i]!=NULL)
// 				UpC->trgVal[i][j]=trgValE[t];
// 			t++;
// //			cout<<i<<","<<j<<endl;
// //			cout<<trgValE[t-1]<<endl;
// 		}
// 	}
// }


void make_ds_up(int *srcBox,upComp_t *UpC) {	//TODO
	int start=0;
	int t=0;
	int size;
	for(int i=0;i<UpC->numSrcBox;i++) {
		srcBox[t++]=start;
		size=UpC->srcBoxSize[i];
		srcBox[t++]=size;
		start+=size;
	}
}

void gpu_up(upComp_t *UpC) {
  int srcDOF, trgDOF;
  GPU_MSG ("Upward computation");
  if (!UpC || !UpC->numSrcBox) { GPU_MSG ("==> No source boxes; skipping..."); return; }
  //	hipSetDevice(0);
//	unsigned int timer;
//	float ms;
//	cutCreateTimer(&timer);

	float *src_dp,*trgVal_dp,*trgCtr_dp,*trgRad_dp;
	int *srcBox_dp;

	// float trgValE[UpC->trgDim*UpC->numSrcBox];
	int srcBox[2*UpC->numSrcBox];

	make_ds_up(srcBox,UpC);

	switch(UpC->kernel_type)
	{
	  case KNL_LAP_S_U:
	    srcDOF=trgDOF=1;
	    break;
	  case KNL_STK_S_U:
	    srcDOF=trgDOF=3;
	    break;
	  default:
	    MPI_ASSERT(false);
	}

	src_dp = gpu_calloc_float ((UpC->numSrc + BLOCK_HEIGHT) * (UpC->dim+srcDOF));
	trgCtr_dp = gpu_calloc_float (UpC->numSrcBox*3);
	trgRad_dp = gpu_calloc_float (UpC->numSrcBox);
	srcBox_dp = gpu_calloc_int (UpC->numSrcBox*2);
	trgVal_dp = gpu_calloc_float (UpC->trgDim*UpC->numSrcBox*trgDOF);

	gpu_copy_cpu2gpu_float (src_dp, UpC->src_, UpC->numSrc * (UpC->dim+srcDOF));
	gpu_copy_cpu2gpu_float (trgCtr_dp, UpC->trgCtr, UpC->numSrcBox*3);
	gpu_copy_cpu2gpu_float (trgRad_dp, UpC->trgRad, UpC->numSrcBox);
	gpu_copy_cpu2gpu_int (srcBox_dp, srcBox, UpC->numSrcBox*2);

	hipMemcpyToSymbol(HIP_SYMBOL(sampos),UpC->samPosF/*samp*/,sizeof(float)*UpC->trgDim*3); GPU_CE;
	int GRID_WIDTH=(int)ceil((float)UpC->numSrcBox/65535.0F);
	int GRID_HEIGHT=(int)ceil((float)UpC->numSrcBox/(float)GRID_WIDTH);
	dim3 GridDim(GRID_HEIGHT, GRID_WIDTH);
//	cout<<"Width: "<<GRID_WIDTH<<" HEIGHT: "<<GRID_HEIGHT<<endl;

	switch(UpC->kernel_type)
	{
	  case KNL_LAP_S_U:
	    if(UpC->trgDim==296) {
	      up_kernel<<<GridDim,BLOCK_HEIGHT>>>(src_dp,trgVal_dp,trgCtr_dp,trgRad_dp,srcBox_dp,UpC->numSrcBox);
	    }
	    else if(UpC->trgDim==152) {
	      up_kernel_4<<<GridDim,BLOCK_HEIGHT>>>(src_dp,trgVal_dp,trgCtr_dp,trgRad_dp,srcBox_dp,UpC->numSrcBox);
	    }
	    else
	    {
	      GPU_MSG ("Upward computations not implemented for this kernel and this accuracy"); //Exit the process?
	      MPI_ASSERT(false);
	    }
	    //also, a generic call can be put here
	    break;
	  case KNL_STK_S_U:
	    if(UpC->trgDim==152)
	      up_kernel_stokes_velocity_4<<<GridDim,BLOCK_HEIGHT>>>(src_dp,trgVal_dp,trgCtr_dp,trgRad_dp,srcBox_dp,UpC->numSrcBox);
	    else
	    {
	      GPU_MSG ("Upward computations not implemented for this kernel and this accuracy"); //Exit the process?
	      MPI_ASSERT(false);
	    }
	    break;
	  default:
	    MPI_ASSERT(false);
	}
	GPU_CE;

	gpu_copy_gpu2cpu_float (UpC->trgVal, trgVal_dp, UpC->trgDim*UpC->numSrcBox*trgDOF);
//	CUT_SAFE_CALL(cutStopTimer(timer));
//	ms = cutGetTimerValue(timer);
//	cout<<"Up kernel: "<<ms<<"ms"<<endl;
	// unmake_ds_up(trgValE,UpC);	//FIXME: copies the gpu output into the 2d array used by the interface... make the interface use a 1d array

	hipFree(src_dp); GPU_CE;
	hipFree(trgCtr_dp); GPU_CE;
	hipFree(trgRad_dp); GPU_CE;
	hipFree(srcBox_dp); GPU_CE;
	hipFree(trgVal_dp); GPU_CE;
}

//////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////

void make_ds_down(int *trgBox,dnComp_t *DnC) {
	int tt=0;
	int tot=0;
	for(int i=0;i<DnC->numTrgBox;i++) {
		int rem=DnC->trgBoxSize[i];
		while(rem>0) {
			trgBox[tt++]=tot;		//start
			int size=(rem<BLOCK_HEIGHT)?rem:BLOCK_HEIGHT;
			trgBox[tt++]=size;		//size
			trgBox[tt++]=i;			//box
			tot+=size;
			rem-=size;
		}
	}
}

// void unmake_ds_down(float *trgValE,dnComp_t *DnC) {
// 	int t=0;
// 	for(int i=0;i<DnC->numTrgBox;i++) {
// 		for(int j=0;j<DnC->trgBoxSize[i];j++) {
// 			if(DnC->trgVal[i]!=NULL) {
// 				DnC->trgVal[i][j]=trgValE[t++];
// //				cout<<DnC->trgVal[i][j]<<endl;
// 			}
// 		}
// 	}
// }

__global__ void dn_kernel(float *trg_dp,float *trgVal_dp,float *srcCtr_dp,float *srcRad_dp,int *trgBox_dp,float *srcDen_dp,int numAugTrg) {
	__shared__ float4 s_sh[64];
	int3 trgBox;

	int uniqueBlockId=blockIdx.y * gridDim.x + blockIdx.x;
	if(uniqueBlockId<numAugTrg) {
		trgBox=((int3*)trgBox_dp)[uniqueBlockId];		//start,size,box

		float3 t_reg=((float3*)trg_dp)[trgBox.x+threadIdx.x];

		float3 srcCtr=((float3*)srcCtr_dp)[trgBox.z];
		float srcRad=srcRad_dp[trgBox.z];

		float dX_reg,dY_reg,dZ_reg;
		float tv_reg=0.0;

		//every thread computes a single src body


		s_sh[threadIdx.x].x=srcCtr.x+srcRad*samposDn[threadIdx.x].x;
		s_sh[threadIdx.x].y=srcCtr.y+srcRad*samposDn[threadIdx.x].y;
		s_sh[threadIdx.x].z=srcCtr.z+srcRad*samposDn[threadIdx.x].z;

		s_sh[threadIdx.x].w=srcDen_dp[152*trgBox.z+threadIdx.x];

		__syncthreads();
		for(int src=0;src<64;src++) {
			dX_reg=s_sh[src].x-t_reg.x;
			dY_reg=s_sh[src].y-t_reg.y;
			dZ_reg=s_sh[src].z-t_reg.z;

			dX_reg*=dX_reg;
			dY_reg*=dY_reg;
			dZ_reg*=dZ_reg;

			dX_reg += dY_reg+dZ_reg;

			dX_reg = rsqrtf(dX_reg);
//@@
dX_reg = dX_reg + (dX_reg-dX_reg);
dX_reg = fmaxf(dX_reg,0.0F);

			tv_reg+=dX_reg*s_sh[src].w;
		}
		__syncthreads();
		s_sh[threadIdx.x].x=srcCtr.x+srcRad*samposDn[64+threadIdx.x].x;
		s_sh[threadIdx.x].y=srcCtr.y+srcRad*samposDn[64+threadIdx.x].y;
		s_sh[threadIdx.x].z=srcCtr.z+srcRad*samposDn[64+threadIdx.x].z;

		s_sh[threadIdx.x].w=srcDen_dp[152*trgBox.z+threadIdx.x+64];

		__syncthreads();
		for(int src=0;src<64;src++) {
			dX_reg=s_sh[src].x-t_reg.x;
			dY_reg=s_sh[src].y-t_reg.y;
			dZ_reg=s_sh[src].z-t_reg.z;

			dX_reg*=dX_reg;
			dY_reg*=dY_reg;
			dZ_reg*=dZ_reg;

			dX_reg += dY_reg+dZ_reg;

			dX_reg = rsqrtf(dX_reg);
//@@
dX_reg = dX_reg + (dX_reg-dX_reg);
dX_reg = fmaxf(dX_reg,0.0F);

			tv_reg+=dX_reg*s_sh[src].w;
		}
		__syncthreads();
		if(threadIdx.x<24) {
			s_sh[threadIdx.x].x=srcCtr.x+srcRad*samposDn[128+threadIdx.x].x;
			s_sh[threadIdx.x].y=srcCtr.y+srcRad*samposDn[128+threadIdx.x].y;
			s_sh[threadIdx.x].z=srcCtr.z+srcRad*samposDn[128+threadIdx.x].z;

			s_sh[threadIdx.x].w=srcDen_dp[152*trgBox.z+threadIdx.x+128];
		}

		__syncthreads();
		for(int src=0;src<24;src++) {
			dX_reg=s_sh[src].x-t_reg.x;
			dY_reg=s_sh[src].y-t_reg.y;
			dZ_reg=s_sh[src].z-t_reg.z;

			dX_reg*=dX_reg;
			dY_reg*=dY_reg;
			dZ_reg*=dZ_reg;

			dX_reg += dY_reg+dZ_reg;

			dX_reg = rsqrtf(dX_reg);
//@@
dX_reg = dX_reg + (dX_reg-dX_reg);
dX_reg = fmaxf(dX_reg,0.0F);

			tv_reg+=dX_reg*s_sh[src].w;
		}

		if(threadIdx.x<trgBox.y)
			trgVal_dp[trgBox.x+threadIdx.x]=tv_reg*PI_4I;
//			trgVal_dp[trgBox.x+threadIdx.x]=trgBox.z;
	}//extra padding block

}

__global__ void dn_kernel_4(float *trg_dp,float *trgVal_dp,float *srcCtr_dp,float *srcRad_dp,int *trgBox_dp,float* srcDen_dp,int numAugTrg) {

	__shared__ float4 s_sh[56];
	int3 trgBox;

	int uniqueBlockId=blockIdx.y * gridDim.x + blockIdx.x;
	if(uniqueBlockId<numAugTrg) {
		trgBox=((int3*)trgBox_dp)[uniqueBlockId];		//start,size,box

		float3 t_reg=((float3*)trg_dp)[trgBox.x+threadIdx.x];

		float3 srcCtr=((float3*)srcCtr_dp)[trgBox.z];
		float srcRad=srcRad_dp[trgBox.z];

		float dX_reg,dY_reg,dZ_reg;
		float tv_reg=0.0;

		//every thread computes a single src body

		if(threadIdx.x<56) {	//no segfaults here

			s_sh[threadIdx.x].x=srcCtr.x+srcRad*samposDn[threadIdx.x].x;
			s_sh[threadIdx.x].y=srcCtr.y+srcRad*samposDn[threadIdx.x].y;
			s_sh[threadIdx.x].z=srcCtr.z+srcRad*samposDn[threadIdx.x].z;

			s_sh[threadIdx.x].w=srcDen_dp[56*trgBox.z+threadIdx.x];
		}
		__syncthreads();
		for(int src=0;src<56;src++) {
			dX_reg=s_sh[src].x-t_reg.x;

			dY_reg=s_sh[src].y-t_reg.y;

			dZ_reg=s_sh[src].z-t_reg.z;

			dX_reg*=dX_reg;
			dY_reg*=dY_reg;
			dZ_reg*=dZ_reg;

			dX_reg += dY_reg+dZ_reg;

			dX_reg = rsqrtf(dX_reg);
//@@
dX_reg = dX_reg + (dX_reg-dX_reg);
dX_reg = fmaxf(dX_reg,0.0F);

			tv_reg+=dX_reg*s_sh[src].w;
		}

		if(threadIdx.x<trgBox.y)
			trgVal_dp[trgBox.x+threadIdx.x]=tv_reg*PI_4I;
	}//extra padding block

}

__global__ void dn_kernel_stokes_fmm_4(float *trg_dp,float *trgVal_dp,float *srcCtr_dp,float *srcRad_dp,int *trgBox_dp,float* srcDen_dp,int numAugTrg)
{

  __shared__ float3 sc_sh[56];
  __shared__ float4 sd_sh[56];
  int3 trgBox;

  int uniqueBlockId=blockIdx.y * gridDim.x + blockIdx.x;
  if(uniqueBlockId<numAugTrg) {
    trgBox=((int3*)trgBox_dp)[uniqueBlockId];		//start,size,box

    float3 t_reg=((float3*)trg_dp)[trgBox.x+threadIdx.x];

    float3 srcCtr=((float3*)srcCtr_dp)[trgBox.z];
    float srcRad=srcRad_dp[trgBox.z];

    float dX_reg,dY_reg,dZ_reg;
    float3 tv_reg={0.0F,0.0F,0.0F} ;

    //every thread computes a single src body

    if(threadIdx.x<56) {	//no segfaults here

      sc_sh[threadIdx.x].x=srcCtr.x+srcRad*samposDn[threadIdx.x].x;
      sc_sh[threadIdx.x].y=srcCtr.y+srcRad*samposDn[threadIdx.x].y;
      sc_sh[threadIdx.x].z=srcCtr.z+srcRad*samposDn[threadIdx.x].z;
      sd_sh[threadIdx.x]=((float4*)srcDen_dp)[56*trgBox.z+threadIdx.x];
    }
    __syncthreads();
    for(int src=0;src<56;src++) {
      dX_reg=sc_sh[src].x-t_reg.x;
      dY_reg=sc_sh[src].y-t_reg.y;
      dZ_reg=sc_sh[src].z-t_reg.z;

      float invR = rsqrtf(dX_reg*dX_reg+dY_reg*dY_reg+dZ_reg*dZ_reg);

      // following two lines set invR to zero if invR is infinity
      invR = invR + (invR-invR);
      invR = fmaxf(invR, 0.0F);

      float4 cur_pot = sd_sh[src];
      float tmp_scalar = (dX_reg*cur_pot.x + dY_reg*cur_pot.y + dZ_reg*cur_pot.z - 2*cur_pot.w)*invR*invR;
      cur_pot.x += tmp_scalar*dX_reg;
      cur_pot.y += tmp_scalar*dY_reg;
      cur_pot.z += tmp_scalar*dZ_reg;

      tv_reg.x += cur_pot.x*invR;
      tv_reg.y += cur_pot.y*invR;
      tv_reg.z += cur_pot.z*invR;
    }

    if(threadIdx.x<trgBox.y)
    {
      trgVal_dp[3*(trgBox.x+threadIdx.x)]  = tv_reg.x*PI_8I;
      trgVal_dp[3*(trgBox.x+threadIdx.x)+1]= tv_reg.y*PI_8I;
      trgVal_dp[3*(trgBox.x+threadIdx.x)+2]= tv_reg.z*PI_8I;
    }
  }//extra padding block
}


int getnumAugTrg(dnComp_t *DnC) {
	int numAugTrg=0;
	for(int i=0;i<DnC->numTrgBox;i++) {
		numAugTrg+=(int)ceil((float)DnC->trgBoxSize[i]/(float)BLOCK_HEIGHT);
	}
	return numAugTrg;
}

void gpu_down(dnComp_t *DnC) {
  GPU_MSG ("Downward (combine) pass");
	int numAugTrg = getnumAugTrg(DnC);
	if (!numAugTrg) { GPU_MSG ("==> numAugTrg == 0; skipping..."); return; }
	float *trg_dp,*trgVal_dp,*srcCtr_dp,*srcRad_dp,*srcDen_dp;
	int *trgBox_dp;	//has start and size and block
	// int trgBox[3*numAugTrg];
	vector<int> trgBox(3*numAugTrg);
	int srcDOF, trgDOF;

	make_ds_down(&trgBox[0],DnC);

	switch(DnC->kernel_type)
	{
	  case KNL_LAP_S_U:
	    srcDOF=trgDOF=1;
	    break;
	  case KNL_STK_F_U:
	    srcDOF=4;
	    trgDOF=3;
	    break;
	  default:
	    MPI_ASSERT(false);
	}


	trg_dp = gpu_calloc_float ((DnC->numTrg+BLOCK_HEIGHT) * (DnC->dim));
	srcCtr_dp = gpu_calloc_float (DnC->numTrgBox*3);
	srcRad_dp = gpu_calloc_float (DnC->numTrgBox);
	trgBox_dp = gpu_calloc_int (numAugTrg*3);
	trgVal_dp = gpu_calloc_float (DnC->numTrg*trgDOF);
	srcDen_dp = gpu_calloc_float (DnC->numTrgBox*DnC->srcDim*srcDOF);

	gpu_copy_cpu2gpu_float (trg_dp, DnC->trg_, DnC->numTrg * DnC->dim);
	gpu_copy_cpu2gpu_float (srcCtr_dp, DnC->srcCtr, DnC->numTrgBox*3);
	gpu_copy_cpu2gpu_float (srcRad_dp, DnC->srcRad, DnC->numTrgBox);
	gpu_copy_cpu2gpu_int (trgBox_dp, &trgBox[0], numAugTrg*3);
	gpu_copy_cpu2gpu_float (srcDen_dp, DnC->srcDen, DnC->numTrgBox*DnC->srcDim*srcDOF);
	hipMemcpyToSymbol(HIP_SYMBOL(samposDn), DnC->samPosF, sizeof(float)*DnC->srcDim*3); GPU_CE;
//	int GRID_HEIGHT=UpC->numSrcBox;
	int GRID_WIDTH=(int)ceil((float)numAugTrg/65535.0F);
	int GRID_HEIGHT=(int)ceil((float)numAugTrg/(float)GRID_WIDTH);
	dim3 GridDim(GRID_HEIGHT, GRID_WIDTH);
//	cout<<"Width: "<<GRID_WIDTH<<" HEIGHT: "<<GRID_HEIGHT<<endl;
	switch (DnC->kernel_type)
	{
	  case KNL_LAP_S_U:
	    if(DnC->srcDim==152) {
	      dn_kernel<<<GridDim,BLOCK_HEIGHT>>>(trg_dp,trgVal_dp,srcCtr_dp,srcRad_dp,trgBox_dp,srcDen_dp,numAugTrg);
	    }
	    else if(DnC->srcDim==56) {
	      dn_kernel_4<<<GridDim,BLOCK_HEIGHT>>>(trg_dp,trgVal_dp,srcCtr_dp,srcRad_dp,trgBox_dp,srcDen_dp,numAugTrg);
	    }
	    else
	    {
	      GPU_MSG ("Downward computations not implemented for this accuracy");	//Exit the process?
	      MPI_ASSERT(false);
	    }
	    GPU_CE;
	    break;
	  case KNL_STK_F_U:
	    if(DnC->srcDim==56) 
	      dn_kernel_stokes_fmm_4<<<GridDim,BLOCK_HEIGHT>>>(trg_dp,trgVal_dp,srcCtr_dp,srcRad_dp,trgBox_dp,srcDen_dp,numAugTrg);
	    else
	    {
	      GPU_MSG ("Downward computations not implemented for this accuracy");
	      MPI_ASSERT(false);
	    }
	    GPU_CE;
	    break;
	  default:
	    MPI_ASSERT(false);
	}


	gpu_copy_gpu2cpu_float (DnC->trgVal, trgVal_dp, DnC->numTrg*trgDOF);

	hipFree(trg_dp); GPU_CE;
	hipFree(srcCtr_dp); GPU_CE;
	hipFree(srcRad_dp); GPU_CE;
	hipFree(trgBox_dp); GPU_CE;
	hipFree(trgVal_dp); GPU_CE;
	hipFree(srcDen_dp); GPU_CE;
}
