#include "hip/hip_runtime.h"
/* Parallel Kernel Independent Fast Multipole Method
   Copyright (C) 2010 George Biros, Harper Langston, Ilya Lashuk
   Copyright (C) 2010, Aparna Chandramowlishwaran, Aashay Shingrapure, Rich Vuduc

This program is free software; you can redistribute it and/or modify
it under the terms of the GNU General Public License as published by
the Free Software Foundation; either version 2, or (at your option)
any later version.

This program is distributed in the hope that it will be useful, but WITHOUT
ANY WARRANTY; without even the implied warranty of MERCHANTABILITY or
FITNESS FOR A PARTICULAR PURPOSE.  See the GNU General Public License
for more details.

You should have received a copy of the GNU General Public License
along with this program; see the file COPYING.  If not, write to the Free
Software Foundation, Inc., 59 Temple Place - Suite 330, Boston, MA
02111-1307, USA.  */

#include <mpi.h>

#include <cstdio>
#include <cstring>
#include <cmath>

#include <hip/hip_runtime.h>
#include <cutil.h>

#include "../p3d/point3d.h"
#include "gpu_setup.h"
#include "kernel3d_mpi.hpp"

#define MPI_ASSERT(c)  mpi_assert__ (((long)c), #c, __FILE__, __LINE__)

#define PI_4I 0.079577471F
#define PI_8I 0.0397887358F

#define BLOCK_HEIGHT 64
#define BLOCK_WIDTH 1

static
void
mpi_assert__ (long cond, const char* str_cond, const char* file, size_t line)
{
  if (!cond) {
    int rank;
    char procname[MPI_MAX_PROCESSOR_NAME+1];
    int procnamelen;
    memset (procname, 0, sizeof (procname));
    MPI_Comm_rank (MPI_COMM_WORLD, &rank);
    MPI_Get_processor_name (procname, &procnamelen);
    fprintf (stderr, "*** [%s:%lu--p%d(%s)] ASSERTION FAILURE: %s ***\n",
	     file, (unsigned long)line, rank, procname, str_cond);
    fflush (stderr);
    MPI_Abort (MPI_COMM_WORLD, 1);
  }
}

#if defined (GPU_CERR)
void
gpu_checkerr__stdout (const char* filename, size_t line)
{
  FILE* fp = stdout;
  hipError_t C_E = hipGetLastError ();
  if (C_E) {
    int rank;
    char procname[MPI_MAX_PROCESSOR_NAME+1];
    int procnamelen;
    memset (procname, 0, sizeof (procname));
    MPI_Comm_rank (MPI_COMM_WORLD, &rank);
    MPI_Get_processor_name (procname, &procnamelen);
    fprintf ((fp), "*** [%s:%lu--p%d(%s)] CUDA ERROR: %s ***\n", filename, line, rank, procname, hipGetErrorString (C_E));
    fflush (fp);
  }
}
#endif

void
gpu_msg__stdout (const char* msg, const char* filename, size_t lineno)
{
  FILE* fp = stdout;
  int rank;
  char procname[MPI_MAX_PROCESSOR_NAME+1];
  int procnamelen;
  memset (procname, 0, sizeof (procname));
  MPI_Comm_rank (MPI_COMM_WORLD, &rank);
  MPI_Get_processor_name (procname, &procnamelen);
  fprintf (fp, "===> [%s:%lu--p%d(%s)] %s\n", filename, lineno, rank, procname, msg);
}

void
gpu_check_pointer (const void* p, const char* fn, size_t l)
{
  if (!p) {
    gpu_msg__stdout ("NULL pointer", fn, l);
    MPI_Abort (MPI_COMM_WORLD, -1);
    MPI_ASSERT (p);
  }
}

size_t
gpu_count (void)
{
 int dev_count;
 CUDA_SAFE_CALL (hipGetDeviceCount (&dev_count)); GPU_CE;
  if (dev_count > 0) {
    fprintf (stderr, "==> Found %d GPU device%s.\n",
   dev_count,
   dev_count == 1 ? "" : "s");
    return (size_t)dev_count;
  }
  return 0; /* no devices found */
}

static
const char *
get_log_dir_ (void)
{
  static const char* log_dir_ = NULL;
  if (!log_dir_) {
    const char* s = getenv ("LOG_DIR");
    if (s && strlen (s) > 0)
      log_dir_ = s;
    else
      log_dir_ = ".";
  }
  MPI_ASSERT (log_dir_);
  return log_dir_;
}

void
gpu_dumpinfo (FILE* fp, size_t dev_id)
{
  FILE* fp_out = fp;
  if (!fp) {
    /* Open 'default' file based on node name */
    int rank = -1;
    char procname[MPI_MAX_PROCESSOR_NAME+1];
    int procnamelen;
    memset (procname, 0, sizeof (procname));
    MPI_Get_processor_name (procname, &procnamelen);
    MPI_Comm_rank (MPI_COMM_WORLD, &rank);

    const char* log_dir = get_log_dir_ ();
    int pathlen = strlen (log_dir) + 1 + MPI_MAX_PROCESSOR_NAME + 15 + 1;
    char* log_file = new char[pathlen];
    MPI_ASSERT (log_file);
    memset (log_file, 0, pathlen);
    sprintf (log_file, "%s/%s--p%d.log", log_dir, procname, rank);
    fp_out = fopen (log_file, "wt");
    delete[] log_file;
    MPI_ASSERT (fp_out);
  }
  hipDeviceProp_t p;
  MPI_ASSERT (dev_id < gpu_count ());
  CUDA_SAFE_CALL(hipGetDeviceProperties(&p, (int)dev_id)); GPU_CE;
  fprintf (fp_out, "==> Device %lu: \"%s\"\n", (unsigned long)dev_id, p.name);
  fprintf (fp_out, " Major revision number: %d\n", p.major);
  fprintf (fp_out, " Minor revision number: %d\n", p.minor);
  fprintf (fp_out, " Total amount of global memory: %u MB\n", p.totalGlobalMem >> 20);
#if CUDART_VERSION >= 2000
  fprintf (fp_out, " Number of multiprocessors: %d\n", p.multiProcessorCount);
  fprintf (fp_out, " Number of cores: %d\n", 8 * p.multiProcessorCount);
#endif
  fprintf (fp_out, " Total amount of constant memory: %u MB\n", p.totalConstMem >> 20);
  fprintf (fp_out, " Total amount of shared memory per block: %u KB\n", p.sharedMemPerBlock >> 10);
  fprintf (fp_out, " Total number of registers available per block: %d\n", p.regsPerBlock);
  fprintf (fp_out, " Warp size: %d\n", p.warpSize);
  fprintf (fp_out, " Maximum number of threads per block: %d\n", p.maxThreadsPerBlock);
  fprintf (fp_out, " Maximum sizes of each dimension of a block: %d x %d x %d\n",
   p.maxThreadsDim[0], p.maxThreadsDim[1], p.maxThreadsDim[2]);
  fprintf (fp_out, " Maximum sizes of each dimension of a grid: %d x %d x %d\n",
   p.maxGridSize[0], p.maxGridSize[1], p.maxGridSize[2]);
  fprintf (fp_out, " Maximum memory pitch: %u bytes\n", p.memPitch);
  fprintf (fp_out, " Texture alignment: %u bytes\n", p.textureAlignment);
  fprintf (fp_out, " Clock rate: %.2f GHz\n", p.clockRate * 1e-6f);
#if CUDART_VERSION >= 2000
  fprintf (fp_out, " Concurrent copy and execution: %s\n", p.deviceOverlap ? "Yes" : "No");
#endif
  if (!fp && fp_out)
    fclose (fp_out);
}

void
gpu_select (size_t dev_id)
{
  fprintf (stderr, "==> Selecting GPU device: %lu\n", (unsigned long)dev_id);
  CUDA_SAFE_CALL (hipSetDevice ((int)dev_id)); GPU_CE;
  gpu_dumpinfo (NULL, dev_id);
}

/** Allocates 'n' bytes, initialized to zero */
void *
gpu_calloc (size_t n)
{
  void* p = NULL;
  if (n) {
    hipMalloc(&p, n); GPU_CE;
    if (!p) {
      int mpirank;
      MPI_Comm_rank (MPI_COMM_WORLD, &mpirank);
      fprintf (stderr, "[%s:%lu::p%d] Can't allocate %lu bytes!\n",
	       __FILE__, __LINE__, mpirank, (unsigned long)n);
    }
    MPI_ASSERT (p);
    hipMemset (p, 0, n); GPU_CE;
  }
  return p;
}

double *
gpu_calloc_double (size_t n)
{
  return (double *)gpu_calloc (n * sizeof (double));
}

float *
gpu_calloc_float (size_t n)
{
  return (float *)gpu_calloc (n * sizeof (float));
}

int *
gpu_calloc_int (size_t n)
{
  return (int *)gpu_calloc (n * sizeof (int));
}


void
gpu_copy_cpu2gpu (void* d, const void* s, size_t n_bytes)
{
  if (n_bytes) {
    hipMemcpy (d, s, n_bytes, hipMemcpyHostToDevice);
    GPU_CE;
  }
}

void
gpu_copy_cpu2gpu_float (float* d, const float* s, size_t n)
{
  gpu_copy_cpu2gpu (d, s, n * sizeof (float));
}

void
gpu_copy_cpu2gpu_double (double* d, const double* s, size_t n)
{
  gpu_copy_cpu2gpu (d, s, n * sizeof (double));
}


void
gpu_copy_cpu2gpu_int (int* d, const int* s, size_t n)
{
  gpu_copy_cpu2gpu (d, s, n * sizeof (int));
}

void
gpu_copy_gpu2cpu (void* d, const void* s, size_t n_bytes)
{
  if (n_bytes) {
    hipMemcpy (d, s, n_bytes, hipMemcpyDeviceToHost);
    GPU_CE;
  }
}

void
gpu_copy_gpu2cpu_float (float* d, const float* s, size_t n)
{
  gpu_copy_gpu2cpu (d, s, n * sizeof (float));
}

void
gpu_copy_gpu2cpu_double (double* d, const double* s, size_t n)
{
  gpu_copy_gpu2cpu (d, s, n * sizeof (double));
}

////////////////////////////////////////BEGIN KERNEL///////////////////////////////////////////////

//#define GRID_WIDTH 1

using namespace std;

#ifdef USE_DOUBLE
__global__ void ulist_kernel(double *t_dp,double *trgVal_dp,
          double *s_dp,
          int *tbdsr_dp,int *tbdsf_dp,int *cs_dp,int *cp_dp,
          int numAugTrg,double kernel_coef) {
  __shared__ double4 s_sh[BLOCK_HEIGHT];
  double3 t_reg;


  int uniqueBlockId=blockIdx.y * gridDim.x + blockIdx.x;
  if(uniqueBlockId<numAugTrg) {


    double tv_reg=0.0F;

    int boxId=tbdsr_dp[uniqueBlockId*3]*2;

    int trgLimit=tbdsr_dp[uniqueBlockId*3+1];
    int trgIdx=tbdsr_dp[uniqueBlockId*3+2]+threadIdx.x;  //can simplify by adding boxid to tbds base to make new pointer

      t_reg=((double3*)t_dp)[trgIdx];


      double dX_reg;
      double dY_reg;
      double dZ_reg;



    int offset_reg=tbdsf_dp[boxId];
    int numSrc_reg=tbdsf_dp[boxId+1];
    int cs_idx_reg=0;

    int *cp_sh=cp_dp+offset_reg;    //TODO: fix this
    int *cs_sh=cs_dp+offset_reg;
    int loc_reg=cp_sh[0]+threadIdx.x;
    int num_thread_reg=threadIdx.x;
    int lastsum=cs_sh[0];

    //fetching cs and cp into shared mem
  //    for(int i=0;i<ceilf((double)numSrcBox_reg/(double)BLOCK_HEIGHT);i++)
  //      if(threadIdx.x<numSrcBox_reg-i*BLOCK_HEIGHT) {
  //        cs_sh[i*BLOCK_HEIGHT+threadIdx.x]=cs_dp[offset_reg+i*BLOCK_HEIGHT+threadIdx.x];
  //        cp_sh[i*BLOCK_HEIGHT+threadIdx.x]=cp_dp[offset_reg+i*BLOCK_HEIGHT+threadIdx.x];
  //      }


    int num_chunk_loop=numSrc_reg/BLOCK_HEIGHT;

    for(int chunk=0;chunk<num_chunk_loop;chunk++) {


      if(num_thread_reg>=lastsum) {
        while(num_thread_reg>=cs_sh[cs_idx_reg]) cs_idx_reg++;
        loc_reg=cp_sh[cs_idx_reg]+(num_thread_reg-cs_sh[cs_idx_reg-1]);
        lastsum=cs_sh[cs_idx_reg];
      }

      __syncthreads();
  #ifdef DS_ORG
      s_sh[threadIdx.x]=((double4*)s_dp)[loc_reg];
  #else
      sx_sh[threadIdx.x]=sx_dp[loc_reg];
      sy_sh[threadIdx.x]=sy_dp[loc_reg];
      sz_sh[threadIdx.x]=sz_dp[loc_reg];
      sd_sh[threadIdx.x]=srcDen_dp[loc_reg];
  #endif

      loc_reg+=BLOCK_HEIGHT;
      num_thread_reg+=BLOCK_HEIGHT;

      __syncthreads();
#pragma unroll 64
      for(int src=0;src<BLOCK_HEIGHT;src++) {
  #ifdef DS_ORG
        dX_reg=s_sh[src].x-t_reg.x;
        dY_reg=s_sh[src].y-t_reg.y;
        dZ_reg=s_sh[src].z-t_reg.z;

        dX_reg*=dX_reg;
        dY_reg*=dY_reg;
        dZ_reg*=dZ_reg;

        dX_reg += dY_reg+dZ_reg;

        dX_reg = rsqrtf(dX_reg);

        dX_reg = dX_reg + (dX_reg-dX_reg);
        dX_reg = fmaxf(dX_reg,0.0F);

        tv_reg+=dX_reg*s_sh[src].w;
  #else
        dX_reg=sx_sh[src]-tx_reg;
        dY_reg=sy_sh[src]-ty_reg;
        dZ_reg=sz_sh[src]-tz_reg;

        dX_reg*=dX_reg;
        dY_reg*=dY_reg;
        dZ_reg*=dZ_reg;

        dX_reg += dY_reg+dZ_reg;

        dX_reg = rsqrtf(dX_reg);

        dX_reg = dX_reg + (dX_reg-dX_reg);
        dX_reg = fmaxf(dX_reg,0.0F);

        tv_reg+=dX_reg*sd_sh[src] ;
  #endif

        }
    } // chunk
    if(num_thread_reg<numSrc_reg) {
      if(num_thread_reg>=lastsum) {
        while(num_thread_reg>=cs_sh[cs_idx_reg]) cs_idx_reg++;
        loc_reg=cp_sh[cs_idx_reg]+(num_thread_reg-cs_sh[cs_idx_reg-1]);
  //      lastsum=cs_sh[cs_idx_reg];
      }
    }
    __syncthreads();
  #ifdef DS_ORG
      s_sh[threadIdx.x]=((double4*)s_dp)[loc_reg];
  #else
      sx_sh[threadIdx.x]=sx_dp[loc_reg];
      sy_sh[threadIdx.x]=sy_dp[loc_reg];
      sz_sh[threadIdx.x]=sz_dp[loc_reg];
      sd_sh[threadIdx.x]=srcDen_dp[loc_reg];
  #endif

    __syncthreads();

    for(int src=0;src<numSrc_reg%BLOCK_HEIGHT;src++) {
  #ifdef DS_ORG
      dX_reg=s_sh[src].x-t_reg.x;
      dY_reg=s_sh[src].y-t_reg.y;
      dZ_reg=s_sh[src].z-t_reg.z;

      dX_reg*=dX_reg;
      dY_reg*=dY_reg;
      dZ_reg*=dZ_reg;

      dX_reg += dY_reg+dZ_reg;

      dX_reg = rsqrtf(dX_reg);
        dX_reg = dX_reg + (dX_reg-dX_reg);
        dX_reg = fmaxf(dX_reg,0.0F);

      tv_reg+=dX_reg*s_sh[src].w;
  #else
      dX_reg=sx_sh[src]-tx_reg;
      dY_reg=sy_sh[src]-ty_reg;
      dZ_reg=sz_sh[src]-tz_reg;

      dX_reg*=dX_reg;
      dY_reg*=dY_reg;
      dZ_reg*=dZ_reg;

      dX_reg += dY_reg+dZ_reg;

      dX_reg = rsqrtf(dX_reg);

      dX_reg = dX_reg + (dX_reg-dX_reg);
      dX_reg = fmaxf(dX_reg,0.0F);

      tv_reg+=dX_reg*sd_sh[src] ;
  #endif

    }


    if(threadIdx.x<trgLimit) {
      trgVal_dp[trgIdx]=tv_reg*PI_4I*kernel_coef;    //div by pi here not inside loop
    }

  }    //extra invalid padding block
}

#else

__global__ void ulist_kernel(float *t_dp,float *trgVal_dp,
          float *s_dp,
          int *tbdsr_dp,int *tbdsf_dp,int *cs_dp,int *cp_dp,
          int numAugTrg,float kernel_coef) {
  __shared__ float4 s_sh[BLOCK_HEIGHT];
  float3 t_reg;


  int uniqueBlockId=blockIdx.y * gridDim.x + blockIdx.x;
  if(uniqueBlockId<numAugTrg) {


    float tv_reg=0.0F;

    int boxId=tbdsr_dp[uniqueBlockId*3]*2;

    int trgLimit=tbdsr_dp[uniqueBlockId*3+1];
    int trgIdx=tbdsr_dp[uniqueBlockId*3+2]+threadIdx.x;  //can simplify by adding boxid to tbds base to make new pointer

      t_reg=((float3*)t_dp)[trgIdx];


      float dX_reg;
      float dY_reg;
      float dZ_reg;



    int offset_reg=tbdsf_dp[boxId];
    int numSrc_reg=tbdsf_dp[boxId+1];
    int cs_idx_reg=0;

    int *cp_sh=cp_dp+offset_reg;    //TODO: fix this
    int *cs_sh=cs_dp+offset_reg;
    int loc_reg=cp_sh[0]+threadIdx.x;
    int num_thread_reg=threadIdx.x;
    int lastsum=cs_sh[0];

    //fetching cs and cp into shared mem
  //    for(int i=0;i<ceilf((float)numSrcBox_reg/(float)BLOCK_HEIGHT);i++)
  //      if(threadIdx.x<numSrcBox_reg-i*BLOCK_HEIGHT) {
  //        cs_sh[i*BLOCK_HEIGHT+threadIdx.x]=cs_dp[offset_reg+i*BLOCK_HEIGHT+threadIdx.x];
  //        cp_sh[i*BLOCK_HEIGHT+threadIdx.x]=cp_dp[offset_reg+i*BLOCK_HEIGHT+threadIdx.x];
  //      }


    int num_chunk_loop=numSrc_reg/BLOCK_HEIGHT;

    for(int chunk=0;chunk<num_chunk_loop;chunk++) {


      if(num_thread_reg>=lastsum) {
        while(num_thread_reg>=cs_sh[cs_idx_reg]) cs_idx_reg++;
        loc_reg=cp_sh[cs_idx_reg]+(num_thread_reg-cs_sh[cs_idx_reg-1]);
        lastsum=cs_sh[cs_idx_reg];
      }

      __syncthreads();
  #ifdef DS_ORG
      s_sh[threadIdx.x]=((float4*)s_dp)[loc_reg];
  #else
      sx_sh[threadIdx.x]=sx_dp[loc_reg];
      sy_sh[threadIdx.x]=sy_dp[loc_reg];
      sz_sh[threadIdx.x]=sz_dp[loc_reg];
      sd_sh[threadIdx.x]=srcDen_dp[loc_reg];
  #endif

      loc_reg+=BLOCK_HEIGHT;
      num_thread_reg+=BLOCK_HEIGHT;

      __syncthreads();
#pragma unroll 64
      for(int src=0;src<BLOCK_HEIGHT;src++) {
  #ifdef DS_ORG
        dX_reg=s_sh[src].x-t_reg.x;
        dY_reg=s_sh[src].y-t_reg.y;
        dZ_reg=s_sh[src].z-t_reg.z;

        dX_reg*=dX_reg;
        dY_reg*=dY_reg;
        dZ_reg*=dZ_reg;

        dX_reg += dY_reg+dZ_reg;

        dX_reg = rsqrtf(dX_reg);

        dX_reg = dX_reg + (dX_reg-dX_reg);
        dX_reg = fmaxf(dX_reg,0.0F);

        tv_reg+=dX_reg*s_sh[src].w;
  #else
        dX_reg=sx_sh[src]-tx_reg;
        dY_reg=sy_sh[src]-ty_reg;
        dZ_reg=sz_sh[src]-tz_reg;

        dX_reg*=dX_reg;
        dY_reg*=dY_reg;
        dZ_reg*=dZ_reg;

        dX_reg += dY_reg+dZ_reg;

        dX_reg = rsqrtf(dX_reg);

        dX_reg = dX_reg + (dX_reg-dX_reg);
        dX_reg = fmaxf(dX_reg,0.0F);

        tv_reg+=dX_reg*sd_sh[src] ;
  #endif

        }
    } // chunk
    if(num_thread_reg<numSrc_reg) {
      if(num_thread_reg>=lastsum) {
        while(num_thread_reg>=cs_sh[cs_idx_reg]) cs_idx_reg++;
        loc_reg=cp_sh[cs_idx_reg]+(num_thread_reg-cs_sh[cs_idx_reg-1]);
  //      lastsum=cs_sh[cs_idx_reg];
      }
    }
    __syncthreads();
  #ifdef DS_ORG
      s_sh[threadIdx.x]=((float4*)s_dp)[loc_reg];
  #else
      sx_sh[threadIdx.x]=sx_dp[loc_reg];
      sy_sh[threadIdx.x]=sy_dp[loc_reg];
      sz_sh[threadIdx.x]=sz_dp[loc_reg];
      sd_sh[threadIdx.x]=srcDen_dp[loc_reg];
  #endif

    __syncthreads();

    for(int src=0;src<numSrc_reg%BLOCK_HEIGHT;src++) {
  #ifdef DS_ORG
      dX_reg=s_sh[src].x-t_reg.x;
      dY_reg=s_sh[src].y-t_reg.y;
      dZ_reg=s_sh[src].z-t_reg.z;

      dX_reg*=dX_reg;
      dY_reg*=dY_reg;
      dZ_reg*=dZ_reg;

      dX_reg += dY_reg+dZ_reg;

      dX_reg = rsqrtf(dX_reg);
        dX_reg = dX_reg + (dX_reg-dX_reg);
        dX_reg = fmaxf(dX_reg,0.0F);

      tv_reg+=dX_reg*s_sh[src].w;
  #else
      dX_reg=sx_sh[src]-tx_reg;
      dY_reg=sy_sh[src]-ty_reg;
      dZ_reg=sz_sh[src]-tz_reg;

      dX_reg*=dX_reg;
      dY_reg*=dY_reg;
      dZ_reg*=dZ_reg;

      dX_reg += dY_reg+dZ_reg;

      dX_reg = rsqrtf(dX_reg);

      dX_reg = dX_reg + (dX_reg-dX_reg);
      dX_reg = fmaxf(dX_reg,0.0F);

      tv_reg+=dX_reg*sd_sh[src] ;
  #endif

    }


    if(threadIdx.x<trgLimit) {
      trgVal_dp[trgIdx]=tv_reg*PI_4I*kernel_coef;    //div by pi here not inside loop
    }

  }    //extra invalid padding block
}

#endif

// kernel for Stokes
#ifdef USE_DOUBLE
__global__ void ulist_kernel_stokes_velocity(double *t_dp,double *trgVal_dp,
    double *s_dp,
    int *tbdsr_dp,int *tbdsf_dp,int *cs_dp,int *cp_dp,
    int numAugTrg, double kernel_coef)
{
  __shared__ double3 sC_sh[BLOCK_HEIGHT]; // sC_sh[i] will contain coordinates of source "i"
  __shared__ double3 sD_sh[BLOCK_HEIGHT]; // sD_sh[i] will contain source density  of source "i" (which has 3 components for Stokes) 

  double3 t_reg;  // position of current target

  int uniqueBlockId=blockIdx.y * gridDim.x + blockIdx.x;

  if(uniqueBlockId<numAugTrg) {


    double3 tv_reg={0.0F,0.0F,0.0F} ;

    int boxId=tbdsr_dp[uniqueBlockId*3]*2;

    int trgLimit=tbdsr_dp[uniqueBlockId*3+1];
    int trgIdx=tbdsr_dp[uniqueBlockId*3+2]+threadIdx.x;  //can simplify by adding boxid to tbds base to make new pointer

    t_reg=((double3*)t_dp)[trgIdx];

    double dX_reg;
    double dY_reg;
    double dZ_reg;


    int offset_reg=tbdsf_dp[boxId];
    int numSrc_reg=tbdsf_dp[boxId+1];
    int cs_idx_reg=0;

    int *cp_sh=cp_dp+offset_reg;    //TODO: fix this
    int *cs_sh=cs_dp+offset_reg;
    int loc_reg=cp_sh[0]+threadIdx.x;
    int num_thread_reg=threadIdx.x;
    int lastsum=cs_sh[0];

    //fetching cs and cp into shared mem
    //    for(int i=0;i<ceilf((double)numSrcBox_reg/(double)BLOCK_HEIGHT);i++)
    //      if(threadIdx.x<numSrcBox_reg-i*BLOCK_HEIGHT) {
    //        cs_sh[i*BLOCK_HEIGHT+threadIdx.x]=cs_dp[offset_reg+i*BLOCK_HEIGHT+threadIdx.x];
    //        cp_sh[i*BLOCK_HEIGHT+threadIdx.x]=cp_dp[offset_reg+i*BLOCK_HEIGHT+threadIdx.x];
    //      }


    int num_chunk_loop=numSrc_reg/BLOCK_HEIGHT;

    for(int chunk=0;chunk<num_chunk_loop;chunk++) {
      if(num_thread_reg>=lastsum) {
	while(num_thread_reg>=cs_sh[cs_idx_reg]) cs_idx_reg++;
	loc_reg=cp_sh[cs_idx_reg]+(num_thread_reg-cs_sh[cs_idx_reg-1]);
	lastsum=cs_sh[cs_idx_reg];
      }

      __syncthreads();
      sC_sh[threadIdx.x]=((double3*)s_dp)[2*loc_reg];
      sD_sh[threadIdx.x]=((double3*)s_dp)[2*loc_reg+1];

      loc_reg+=BLOCK_HEIGHT;
      num_thread_reg+=BLOCK_HEIGHT;

      __syncthreads();
#pragma unroll 32
      for(int src=0;src<BLOCK_HEIGHT;src++) {
	dX_reg=sC_sh[src].x-t_reg.x;
	dY_reg=sC_sh[src].y-t_reg.y;
	dZ_reg=sC_sh[src].z-t_reg.z;

	double invR = rsqrtf(dX_reg*dX_reg+dY_reg*dY_reg+dZ_reg*dZ_reg);

	// following two lines set invR to zero if invR is infinity
	invR = invR + (invR-invR);
	invR = fmaxf(invR, 0.0F);

	double3 cur_pot = sD_sh[src];
	double tmp_scalar = (dX_reg*cur_pot.x + dY_reg*cur_pot.y + dZ_reg*cur_pot.z)*invR*invR;
	cur_pot.x += tmp_scalar*dX_reg;
	cur_pot.y += tmp_scalar*dY_reg;
	cur_pot.z += tmp_scalar*dZ_reg;

	tv_reg.x += cur_pot.x*invR;
	tv_reg.y += cur_pot.y*invR;
	tv_reg.z += cur_pot.z*invR;
      }
    } // chunk

    if(num_thread_reg<numSrc_reg) {
      if(num_thread_reg>=lastsum) {
	while(num_thread_reg>=cs_sh[cs_idx_reg]) cs_idx_reg++;
	loc_reg=cp_sh[cs_idx_reg]+(num_thread_reg-cs_sh[cs_idx_reg-1]);
	//      lastsum=cs_sh[cs_idx_reg];
      }
    }
    __syncthreads();

    sC_sh[threadIdx.x]=((double3*)s_dp)[2*loc_reg];
    sD_sh[threadIdx.x]=((double3*)s_dp)[2*loc_reg+1];

    __syncthreads();

    for(int src=0;src<numSrc_reg%BLOCK_HEIGHT;src++) {
      dX_reg=sC_sh[src].x-t_reg.x;
      dY_reg=sC_sh[src].y-t_reg.y;
      dZ_reg=sC_sh[src].z-t_reg.z;

      double invR = rsqrtf(dX_reg*dX_reg+dY_reg*dY_reg+dZ_reg*dZ_reg);

      // following two lines set invR to zero if invR is infinity
      invR = invR + (invR-invR);
      invR = fmaxf(invR, 0.0F);

      double3 cur_pot = sD_sh[src];
      double tmp_scalar = (dX_reg*cur_pot.x + dY_reg*cur_pot.y + dZ_reg*cur_pot.z)*invR*invR;
      cur_pot.x += tmp_scalar*dX_reg;
      cur_pot.y += tmp_scalar*dY_reg;
      cur_pot.z += tmp_scalar*dZ_reg;

      tv_reg.x += cur_pot.x*invR;
      tv_reg.y += cur_pot.y*invR;
      tv_reg.z += cur_pot.z*invR;
    }


    if(threadIdx.x<trgLimit) {
      trgVal_dp[3*trgIdx]   = tv_reg.x*PI_8I*kernel_coef;    //div by pi here not inside loop
      trgVal_dp[3*trgIdx+1] = tv_reg.y*PI_8I*kernel_coef;
      trgVal_dp[3*trgIdx+2] = tv_reg.z*PI_8I*kernel_coef;
    }
  }    //extra invalid padding block -- what  ?????????
}

#else

__global__ void ulist_kernel_stokes_velocity(float *t_dp,float *trgVal_dp,
    float *s_dp,
    int *tbdsr_dp,int *tbdsf_dp,int *cs_dp,int *cp_dp,
    int numAugTrg, float kernel_coef)
{
  __shared__ float3 sC_sh[BLOCK_HEIGHT]; // sC_sh[i] will contain coordinates of source "i"
  __shared__ float3 sD_sh[BLOCK_HEIGHT]; // sD_sh[i] will contain source density  of source "i" (which has 3 components for Stokes) 

  float3 t_reg;  // position of current target

  int uniqueBlockId=blockIdx.y * gridDim.x + blockIdx.x;

  if(uniqueBlockId<numAugTrg) {


    float3 tv_reg={0.0F,0.0F,0.0F} ;

    int boxId=tbdsr_dp[uniqueBlockId*3]*2;

    int trgLimit=tbdsr_dp[uniqueBlockId*3+1];
    int trgIdx=tbdsr_dp[uniqueBlockId*3+2]+threadIdx.x;  //can simplify by adding boxid to tbds base to make new pointer

    t_reg=((float3*)t_dp)[trgIdx];

    float dX_reg;
    float dY_reg;
    float dZ_reg;


    int offset_reg=tbdsf_dp[boxId];
    int numSrc_reg=tbdsf_dp[boxId+1];
    int cs_idx_reg=0;

    int *cp_sh=cp_dp+offset_reg;    //TODO: fix this
    int *cs_sh=cs_dp+offset_reg;
    int loc_reg=cp_sh[0]+threadIdx.x;
    int num_thread_reg=threadIdx.x;
    int lastsum=cs_sh[0];

    //fetching cs and cp into shared mem
    //    for(int i=0;i<ceilf((float)numSrcBox_reg/(float)BLOCK_HEIGHT);i++)
    //      if(threadIdx.x<numSrcBox_reg-i*BLOCK_HEIGHT) {
    //        cs_sh[i*BLOCK_HEIGHT+threadIdx.x]=cs_dp[offset_reg+i*BLOCK_HEIGHT+threadIdx.x];
    //        cp_sh[i*BLOCK_HEIGHT+threadIdx.x]=cp_dp[offset_reg+i*BLOCK_HEIGHT+threadIdx.x];
    //      }


    int num_chunk_loop=numSrc_reg/BLOCK_HEIGHT;

    for(int chunk=0;chunk<num_chunk_loop;chunk++) {
      if(num_thread_reg>=lastsum) {
	while(num_thread_reg>=cs_sh[cs_idx_reg]) cs_idx_reg++;
	loc_reg=cp_sh[cs_idx_reg]+(num_thread_reg-cs_sh[cs_idx_reg-1]);
	lastsum=cs_sh[cs_idx_reg];
      }

      __syncthreads();
      sC_sh[threadIdx.x]=((float3*)s_dp)[2*loc_reg];
      sD_sh[threadIdx.x]=((float3*)s_dp)[2*loc_reg+1];

      loc_reg+=BLOCK_HEIGHT;
      num_thread_reg+=BLOCK_HEIGHT;

      __syncthreads();
#pragma unroll 32
      for(int src=0;src<BLOCK_HEIGHT;src++) {
	dX_reg=sC_sh[src].x-t_reg.x;
	dY_reg=sC_sh[src].y-t_reg.y;
	dZ_reg=sC_sh[src].z-t_reg.z;

	float invR = rsqrtf(dX_reg*dX_reg+dY_reg*dY_reg+dZ_reg*dZ_reg);

	// following two lines set invR to zero if invR is infinity
	invR = invR + (invR-invR);
	invR = fmaxf(invR, 0.0F);

	float3 cur_pot = sD_sh[src];
	float tmp_scalar = (dX_reg*cur_pot.x + dY_reg*cur_pot.y + dZ_reg*cur_pot.z)*invR*invR;
	cur_pot.x += tmp_scalar*dX_reg;
	cur_pot.y += tmp_scalar*dY_reg;
	cur_pot.z += tmp_scalar*dZ_reg;

	tv_reg.x += cur_pot.x*invR;
	tv_reg.y += cur_pot.y*invR;
	tv_reg.z += cur_pot.z*invR;
      }
    } // chunk

    if(num_thread_reg<numSrc_reg) {
      if(num_thread_reg>=lastsum) {
	while(num_thread_reg>=cs_sh[cs_idx_reg]) cs_idx_reg++;
	loc_reg=cp_sh[cs_idx_reg]+(num_thread_reg-cs_sh[cs_idx_reg-1]);
	//      lastsum=cs_sh[cs_idx_reg];
      }
    }
    __syncthreads();

    sC_sh[threadIdx.x]=((float3*)s_dp)[2*loc_reg];
    sD_sh[threadIdx.x]=((float3*)s_dp)[2*loc_reg+1];

    __syncthreads();

    for(int src=0;src<numSrc_reg%BLOCK_HEIGHT;src++) {
      dX_reg=sC_sh[src].x-t_reg.x;
      dY_reg=sC_sh[src].y-t_reg.y;
      dZ_reg=sC_sh[src].z-t_reg.z;

      float invR = rsqrtf(dX_reg*dX_reg+dY_reg*dY_reg+dZ_reg*dZ_reg);

      // following two lines set invR to zero if invR is infinity
      invR = invR + (invR-invR);
      invR = fmaxf(invR, 0.0F);

      float3 cur_pot = sD_sh[src];
      float tmp_scalar = (dX_reg*cur_pot.x + dY_reg*cur_pot.y + dZ_reg*cur_pot.z)*invR*invR;
      cur_pot.x += tmp_scalar*dX_reg;
      cur_pot.y += tmp_scalar*dY_reg;
      cur_pot.z += tmp_scalar*dZ_reg;

      tv_reg.x += cur_pot.x*invR;
      tv_reg.y += cur_pot.y*invR;
      tv_reg.z += cur_pot.z*invR;
    }


    if(threadIdx.x<trgLimit) {
      trgVal_dp[3*trgIdx]   = tv_reg.x*PI_8I*kernel_coef;    //div by pi here not inside loop
      trgVal_dp[3*trgIdx+1] = tv_reg.y*PI_8I*kernel_coef;
      trgVal_dp[3*trgIdx+2] = tv_reg.z*PI_8I*kernel_coef;
    }
  }    //extra invalid padding block -- what  ?????????
}

#endif





// special "Stokes FMM" kernel -- used for equivalent densities for Stokes-velocity
#ifdef USE_DOUBLE
__global__ void ulist_kernel_stokes_fmm(double *t_dp,double *trgVal_dp,
    double *s_dp,
    int *tbdsr_dp,int *tbdsf_dp,int *cs_dp,int *cp_dp,
    int numAugTrg, double kernel_coef)
{
  __shared__ double3 sC_sh[BLOCK_HEIGHT]; // sC_sh[i] will contain coordinates of source "i"
  __shared__ double4 sD_sh[BLOCK_HEIGHT]; // sD_sh[i] will contain source density  of source "i" (which has 4 components for Stokes-fmm ) 

  double3 t_reg;  // position of current target

  int uniqueBlockId=blockIdx.y * gridDim.x + blockIdx.x;

  if(uniqueBlockId<numAugTrg) {


    double3 tv_reg={0.0F,0.0F,0.0F} ;

    int boxId=tbdsr_dp[uniqueBlockId*3]*2;

    int trgLimit=tbdsr_dp[uniqueBlockId*3+1];
    int trgIdx=tbdsr_dp[uniqueBlockId*3+2]+threadIdx.x;  //can simplify by adding boxid to tbds base to make new pointer

    t_reg=((double3*)t_dp)[trgIdx];

    double dX_reg;
    double dY_reg;
    double dZ_reg;


    int offset_reg=tbdsf_dp[boxId];
    int numSrc_reg=tbdsf_dp[boxId+1];
    int cs_idx_reg=0;

    int *cp_sh=cp_dp+offset_reg;    //TODO: fix this
    int *cs_sh=cs_dp+offset_reg;
    int loc_reg=cp_sh[0]+threadIdx.x;
    int num_thread_reg=threadIdx.x;
    int lastsum=cs_sh[0];

    int num_chunk_loop=numSrc_reg/BLOCK_HEIGHT;

    for(int chunk=0;chunk<num_chunk_loop;chunk++) {
      if(num_thread_reg>=lastsum) {
	while(num_thread_reg>=cs_sh[cs_idx_reg]) cs_idx_reg++;
	loc_reg=cp_sh[cs_idx_reg]+(num_thread_reg-cs_sh[cs_idx_reg-1]);
	lastsum=cs_sh[cs_idx_reg];
      }

      __syncthreads();
      sC_sh[threadIdx.x].x=s_dp[7*loc_reg];
      sC_sh[threadIdx.x].y=s_dp[7*loc_reg+1];
      sC_sh[threadIdx.x].z=s_dp[7*loc_reg+2];

      sD_sh[threadIdx.x].x=s_dp[7*loc_reg+3];
      sD_sh[threadIdx.x].y=s_dp[7*loc_reg+4];
      sD_sh[threadIdx.x].z=s_dp[7*loc_reg+5];
      sD_sh[threadIdx.x].w=s_dp[7*loc_reg+6];

      loc_reg+=BLOCK_HEIGHT;
      num_thread_reg+=BLOCK_HEIGHT;

      __syncthreads();
#pragma unroll 32
      for(int src=0;src<BLOCK_HEIGHT;src++) {
	dX_reg=sC_sh[src].x-t_reg.x;
	dY_reg=sC_sh[src].y-t_reg.y;
	dZ_reg=sC_sh[src].z-t_reg.z;

	double invR = rsqrtf(dX_reg*dX_reg+dY_reg*dY_reg+dZ_reg*dZ_reg);

	// following two lines set invR to zero if invR is infinity
	invR = invR + (invR-invR);
	invR = fmaxf(invR, 0.0F);

	double4 cur_pot = sD_sh[src];
	double tmp_scalar = (dX_reg*cur_pot.x + dY_reg*cur_pot.y + dZ_reg*cur_pot.z - 2/kernel_coef*cur_pot.w)*invR*invR;
	cur_pot.x += tmp_scalar*dX_reg;
	cur_pot.y += tmp_scalar*dY_reg;
	cur_pot.z += tmp_scalar*dZ_reg;

	tv_reg.x += cur_pot.x*invR;
	tv_reg.y += cur_pot.y*invR;
	tv_reg.z += cur_pot.z*invR;
      }
    } // chunk

    if(num_thread_reg<numSrc_reg) {
      if(num_thread_reg>=lastsum) {
	while(num_thread_reg>=cs_sh[cs_idx_reg]) cs_idx_reg++;
	loc_reg=cp_sh[cs_idx_reg]+(num_thread_reg-cs_sh[cs_idx_reg-1]);
	//      lastsum=cs_sh[cs_idx_reg];
      }
    }
    __syncthreads();

    sC_sh[threadIdx.x].x=s_dp[7*loc_reg];
    sC_sh[threadIdx.x].y=s_dp[7*loc_reg+1];
    sC_sh[threadIdx.x].z=s_dp[7*loc_reg+2];

    sD_sh[threadIdx.x].x=s_dp[7*loc_reg+3];
    sD_sh[threadIdx.x].y=s_dp[7*loc_reg+4];
    sD_sh[threadIdx.x].z=s_dp[7*loc_reg+5];
    sD_sh[threadIdx.x].w=s_dp[7*loc_reg+6];

    __syncthreads();

    for(int src=0;src<numSrc_reg%BLOCK_HEIGHT;src++) {
      dX_reg=sC_sh[src].x-t_reg.x;
      dY_reg=sC_sh[src].y-t_reg.y;
      dZ_reg=sC_sh[src].z-t_reg.z;

      double invR = rsqrtf(dX_reg*dX_reg+dY_reg*dY_reg+dZ_reg*dZ_reg);

      // following two lines set invR to zero if invR is infinity
      invR = invR + (invR-invR);
      invR = fmaxf(invR, 0.0F);

      double4 cur_pot = sD_sh[src];
      double tmp_scalar = (dX_reg*cur_pot.x + dY_reg*cur_pot.y + dZ_reg*cur_pot.z - 2/kernel_coef*cur_pot.w)*invR*invR;
      cur_pot.x += tmp_scalar*dX_reg;
      cur_pot.y += tmp_scalar*dY_reg;
      cur_pot.z += tmp_scalar*dZ_reg;

      tv_reg.x += cur_pot.x*invR;
      tv_reg.y += cur_pot.y*invR;
      tv_reg.z += cur_pot.z*invR;
    }


    if(threadIdx.x<trgLimit) {
      trgVal_dp[3*trgIdx]   = tv_reg.x*PI_8I*kernel_coef;    //div by pi here not inside loop
      trgVal_dp[3*trgIdx+1] = tv_reg.y*PI_8I*kernel_coef;
      trgVal_dp[3*trgIdx+2] = tv_reg.z*PI_8I*kernel_coef;
    }
  }    //extra invalid padding block -- what  ?????????
}

#else

__global__ void ulist_kernel_stokes_fmm(float *t_dp,float *trgVal_dp,
    float *s_dp,
    int *tbdsr_dp,int *tbdsf_dp,int *cs_dp,int *cp_dp,
    int numAugTrg, float kernel_coef)
{
  __shared__ float3 sC_sh[BLOCK_HEIGHT]; // sC_sh[i] will contain coordinates of source "i"
  __shared__ float4 sD_sh[BLOCK_HEIGHT]; // sD_sh[i] will contain source density  of source "i" (which has 4 components for Stokes-fmm ) 

  float3 t_reg;  // position of current target

  int uniqueBlockId=blockIdx.y * gridDim.x + blockIdx.x;

  if(uniqueBlockId<numAugTrg) {


    float3 tv_reg={0.0F,0.0F,0.0F} ;

    int boxId=tbdsr_dp[uniqueBlockId*3]*2;

    int trgLimit=tbdsr_dp[uniqueBlockId*3+1];
    int trgIdx=tbdsr_dp[uniqueBlockId*3+2]+threadIdx.x;  //can simplify by adding boxid to tbds base to make new pointer

    t_reg=((float3*)t_dp)[trgIdx];

    float dX_reg;
    float dY_reg;
    float dZ_reg;


    int offset_reg=tbdsf_dp[boxId];
    int numSrc_reg=tbdsf_dp[boxId+1];
    int cs_idx_reg=0;

    int *cp_sh=cp_dp+offset_reg;    //TODO: fix this
    int *cs_sh=cs_dp+offset_reg;
    int loc_reg=cp_sh[0]+threadIdx.x;
    int num_thread_reg=threadIdx.x;
    int lastsum=cs_sh[0];

    int num_chunk_loop=numSrc_reg/BLOCK_HEIGHT;

    for(int chunk=0;chunk<num_chunk_loop;chunk++) {
      if(num_thread_reg>=lastsum) {
	while(num_thread_reg>=cs_sh[cs_idx_reg]) cs_idx_reg++;
	loc_reg=cp_sh[cs_idx_reg]+(num_thread_reg-cs_sh[cs_idx_reg-1]);
	lastsum=cs_sh[cs_idx_reg];
      }

      __syncthreads();
      sC_sh[threadIdx.x].x=s_dp[7*loc_reg];
      sC_sh[threadIdx.x].y=s_dp[7*loc_reg+1];
      sC_sh[threadIdx.x].z=s_dp[7*loc_reg+2];

      sD_sh[threadIdx.x].x=s_dp[7*loc_reg+3];
      sD_sh[threadIdx.x].y=s_dp[7*loc_reg+4];
      sD_sh[threadIdx.x].z=s_dp[7*loc_reg+5];
      sD_sh[threadIdx.x].w=s_dp[7*loc_reg+6];

      loc_reg+=BLOCK_HEIGHT;
      num_thread_reg+=BLOCK_HEIGHT;

      __syncthreads();
#pragma unroll 32
      for(int src=0;src<BLOCK_HEIGHT;src++) {
	dX_reg=sC_sh[src].x-t_reg.x;
	dY_reg=sC_sh[src].y-t_reg.y;
	dZ_reg=sC_sh[src].z-t_reg.z;

	float invR = rsqrtf(dX_reg*dX_reg+dY_reg*dY_reg+dZ_reg*dZ_reg);

	// following two lines set invR to zero if invR is infinity
	invR = invR + (invR-invR);
	invR = fmaxf(invR, 0.0F);

	float4 cur_pot = sD_sh[src];
	float tmp_scalar = (dX_reg*cur_pot.x + dY_reg*cur_pot.y + dZ_reg*cur_pot.z - 2/kernel_coef*cur_pot.w)*invR*invR;
	cur_pot.x += tmp_scalar*dX_reg;
	cur_pot.y += tmp_scalar*dY_reg;
	cur_pot.z += tmp_scalar*dZ_reg;

	tv_reg.x += cur_pot.x*invR;
	tv_reg.y += cur_pot.y*invR;
	tv_reg.z += cur_pot.z*invR;
      }
    } // chunk

    if(num_thread_reg<numSrc_reg) {
      if(num_thread_reg>=lastsum) {
	while(num_thread_reg>=cs_sh[cs_idx_reg]) cs_idx_reg++;
	loc_reg=cp_sh[cs_idx_reg]+(num_thread_reg-cs_sh[cs_idx_reg-1]);
	//      lastsum=cs_sh[cs_idx_reg];
      }
    }
    __syncthreads();

    sC_sh[threadIdx.x].x=s_dp[7*loc_reg];
    sC_sh[threadIdx.x].y=s_dp[7*loc_reg+1];
    sC_sh[threadIdx.x].z=s_dp[7*loc_reg+2];

    sD_sh[threadIdx.x].x=s_dp[7*loc_reg+3];
    sD_sh[threadIdx.x].y=s_dp[7*loc_reg+4];
    sD_sh[threadIdx.x].z=s_dp[7*loc_reg+5];
    sD_sh[threadIdx.x].w=s_dp[7*loc_reg+6];

    __syncthreads();

    for(int src=0;src<numSrc_reg%BLOCK_HEIGHT;src++) {
      dX_reg=sC_sh[src].x-t_reg.x;
      dY_reg=sC_sh[src].y-t_reg.y;
      dZ_reg=sC_sh[src].z-t_reg.z;

      float invR = rsqrtf(dX_reg*dX_reg+dY_reg*dY_reg+dZ_reg*dZ_reg);

      // following two lines set invR to zero if invR is infinity
      invR = invR + (invR-invR);
      invR = fmaxf(invR, 0.0F);

      float4 cur_pot = sD_sh[src];
      float tmp_scalar = (dX_reg*cur_pot.x + dY_reg*cur_pot.y + dZ_reg*cur_pot.z - 2/kernel_coef*cur_pot.w)*invR*invR;
      cur_pot.x += tmp_scalar*dX_reg;
      cur_pot.y += tmp_scalar*dY_reg;
      cur_pot.z += tmp_scalar*dZ_reg;

      tv_reg.x += cur_pot.x*invR;
      tv_reg.y += cur_pot.y*invR;
      tv_reg.z += cur_pot.z*invR;
    }


    if(threadIdx.x<trgLimit) {
      trgVal_dp[3*trgIdx]   = tv_reg.x*PI_8I*kernel_coef;    //div by pi here not inside loop
      trgVal_dp[3*trgIdx+1] = tv_reg.y*PI_8I*kernel_coef;
      trgVal_dp[3*trgIdx+2] = tv_reg.z*PI_8I*kernel_coef;
    }
  }    //extra invalid padding block -- what  ?????????
}

#endif

void make_ds(int **tbdsf, int **tbdsr, int **cs, int **cp, point3d_t* P,int *numAugTrg,int *numSrcBoxTot) {
  for(int i=0;i<P->numTrgBox;i++) {
    *numAugTrg+=(P->trgBoxSize[i]/BLOCK_HEIGHT+((P->trgBoxSize[i]%BLOCK_HEIGHT)?1:0));
    *numSrcBoxTot+=P->uListLen[i];

  }
  int srcidx[P->numSrcBox];
  int srcsum=0;
  for(int i=0;i<P->numSrcBox;i++) {
    srcidx[i]=srcsum;
    srcsum+=P->srcBoxSize[i];
  }
//  cout<<"Split "<<P->numTrgBox<<" targets boxes into "<<*numAugTrg<<endl;
//  cout<<"Total source boxes: "<<*numSrcBoxTot<<endl;

  *tbdsf=(int*)malloc(sizeof(int)*2*P->numTrgBox); MPI_ASSERT (*tbdsf || !P->numTrgBox);
  *tbdsr=(int*)malloc(sizeof(int)*3**numAugTrg); MPI_ASSERT (*tbdsr || !numAugTrg);
  *cs=(int*)malloc(sizeof(int)**numSrcBoxTot); MPI_ASSERT (*cs || !numSrcBoxTot);
  *cp=(int*)malloc(sizeof(int)**numSrcBoxTot); MPI_ASSERT (*cp || !numSrcBoxTot);

  int cc=0;
  int tt=0;
  int tbi=0;

  for(int i=0;i<P->numTrgBox;i++) {
    (*tbdsf)[i*2]=cc;
    int cumulSum=0;
    for(int k=0;k<P->uListLen[i];k++) {
      int srcbox=P->uList[i][k];
      cumulSum+=P->srcBoxSize[srcbox];
      (*cs)[cc]=cumulSum;
      (*cp)[cc]=srcidx[srcbox];
      cc++;
    }
    (*tbdsf)[i*2+1]=cumulSum;
    int remtrg=P->trgBoxSize[i];
    while(remtrg>0) {
      (*tbdsr)[3*tbi]=i;
      (*tbdsr)[3*tbi+1]=(remtrg<BLOCK_HEIGHT)?remtrg:BLOCK_HEIGHT;
      (*tbdsr)[3*tbi+2]=tt;
      tt+=(*tbdsr)[3*tbi+1];
      tbi++;    //tbi corresponds to gpu block id
      remtrg-=BLOCK_HEIGHT;
    }
  }
}

//extern "C"
//{

#ifdef USE_DOUBLE
void dense_inter_gpu(point3d_t *P) {
  double *s_dp,*t_dp;
  double *trgVal_dp;
  int *tbdsf_dp, *tbdsr_dp;
  int *tbdsf,*tbdsr,*cs,*cp,numAugTrg=0,numSrcBoxTot=0;
  int *cs_dp,*cp_dp;
  int srcDOF, trgDOF;

  GPU_MSG ("GPU U-list");

  make_ds (&tbdsf, &tbdsr, &cs, &cp, P, &numAugTrg, &numSrcBoxTot);

  switch(P->kernel_type)
  {
    case KNL_LAP_S_U:
      srcDOF=trgDOF=1;
      break;
    case KNL_STK_S_U:
      srcDOF=trgDOF=3;
      break;
    case KNL_STK_F_U:
      srcDOF=4;
      trgDOF=3;
      break;
    default:
      MPI_ASSERT(false);
  }

  hipMalloc((void**)&s_dp,(P->numSrc + BLOCK_HEIGHT) * (3+srcDOF)*sizeof(double));
  // s_dp = gpu_calloc_double ((P->numSrc + BLOCK_HEIGHT) * (3+srcDOF)); /* Padded by BLOCK_HEIGHT */

  hipMalloc((void**)&t_dp,(P->numTrg + BLOCK_HEIGHT) * 3*sizeof(double));
  // t_dp = gpu_calloc_double ((P->numTrg + BLOCK_HEIGHT) * 3);

  // trgVal_dp = gpu_calloc_double (P->numTrg*trgDOF);
  hipMalloc( (void**)&trgVal_dp, P->numTrg*trgDOF*sizeof(double));

  tbdsf_dp = gpu_calloc_int (P->numTrgBox * 2);
  tbdsr_dp = gpu_calloc_int (numAugTrg * 3);
  cs_dp = gpu_calloc_int (numSrcBoxTot);
  cp_dp = gpu_calloc_int (numSrcBoxTot);

 
  //Put data into the device
  gpu_copy_cpu2gpu_double (s_dp, P->src_, P->numSrc * (3+srcDOF));
  gpu_copy_cpu2gpu_double (t_dp, P->trg_, P->numTrg * 3);

  gpu_copy_cpu2gpu_int (tbdsf_dp, tbdsf, 2 * P->numTrgBox);
  gpu_copy_cpu2gpu_int (tbdsr_dp, tbdsr, 3 * numAugTrg);
  gpu_copy_cpu2gpu_int (cs_dp, cs, numSrcBoxTot);
  gpu_copy_cpu2gpu_int (cp_dp, cp, numSrcBoxTot);

  //kernel call
  int GRID_WIDTH=(int)ceil((double)numAugTrg/65535.0F);
  int GRID_HEIGHT=(int)ceil((double)numAugTrg/(double)GRID_WIDTH);
  dim3 BlockDim (BLOCK_HEIGHT,BLOCK_WIDTH);  //Block width will be 1
  dim3 GridDim (GRID_HEIGHT, GRID_WIDTH);    //Grid width should be 1
  //fprintf (stdout, "@@ [%s:%lu::p%d] numAugTrg=%d; BlockDim x GridDim = [%d x %d] x [%d x %d]\n", __FILE__, (unsigned long)__LINE__, mpirank, numAugTrg, BLOCK_HEIGHT, BLOCK_WIDTH, GRID_HEIGHT, GRID_WIDTH);

#if defined (__DEVICE_EMULATION__)
  GPU_MSG (">>> Device emulation mode <<<\n");
#endif
  if (numAugTrg) // No need to call kernel if numAugTrg == 0
  switch(P->kernel_type)
  {
    case KNL_LAP_S_U:
      ulist_kernel<<<GridDim,BLOCK_HEIGHT>>>(t_dp,trgVal_dp,s_dp,tbdsr_dp,tbdsf_dp,cs_dp,cp_dp,numAugTrg,1/P->kernel_coef[0]); GPU_CE;
      break;
    case KNL_STK_S_U:
      ulist_kernel_stokes_velocity<<<GridDim,BLOCK_HEIGHT>>>(t_dp,trgVal_dp,s_dp,tbdsr_dp,tbdsf_dp,cs_dp,cp_dp,numAugTrg,1/P->kernel_coef[0]); GPU_CE;
      break;
    case KNL_STK_F_U:
      ulist_kernel_stokes_fmm<<<GridDim,BLOCK_HEIGHT>>>(t_dp,trgVal_dp,s_dp,tbdsr_dp,tbdsf_dp,cs_dp,cp_dp,numAugTrg,1/P->kernel_coef[0]); GPU_CE;
      break;
    default:
      MPI_ASSERT(false);
  }

  gpu_copy_gpu2cpu_double (P->trgVal, trgVal_dp, P->numTrg*trgDOF);

  hipFree(s_dp); GPU_CE;
  hipFree(t_dp); GPU_CE;

  hipFree(trgVal_dp); GPU_CE;
  hipFree(tbdsf_dp); GPU_CE;
  hipFree(tbdsr_dp); GPU_CE;
  hipFree(cs_dp); GPU_CE;
  hipFree(cp_dp); GPU_CE;

  free(cs);
  free(cp);
  free(tbdsf);
  free(tbdsr);
}

#else

void dense_inter_gpu(point3d_t *P) {
  float *s_dp,*t_dp;
  float *trgVal_dp;
  int *tbdsf_dp, *tbdsr_dp;
  int *tbdsf,*tbdsr,*cs,*cp,numAugTrg=0,numSrcBoxTot=0;
  int *cs_dp,*cp_dp;
  int srcDOF, trgDOF;

  GPU_MSG ("GPU U-list");

  make_ds (&tbdsf, &tbdsr, &cs, &cp, P, &numAugTrg, &numSrcBoxTot);

  switch(P->kernel_type)
  {
    case KNL_LAP_S_U:
      srcDOF=trgDOF=1;
      break;
    case KNL_STK_S_U:
      srcDOF=trgDOF=3;
      break;
    case KNL_STK_F_U:
      srcDOF=4;
      trgDOF=3;
      break;
    default:
      MPI_ASSERT(false);
  }

  hipMalloc((void**)&s_dp,(P->numSrc + BLOCK_HEIGHT) * (3+srcDOF)*sizeof(float));
  // s_dp = gpu_calloc_float ((P->numSrc + BLOCK_HEIGHT) * (3+srcDOF)); /* Padded by BLOCK_HEIGHT */

  hipMalloc((void**)&t_dp,(P->numTrg + BLOCK_HEIGHT) * 3*sizeof(float));
  // t_dp = gpu_calloc_float ((P->numTrg + BLOCK_HEIGHT) * 3);

  // trgVal_dp = gpu_calloc_float (P->numTrg*trgDOF);
  hipMalloc( (void**)&trgVal_dp, P->numTrg*trgDOF*sizeof(float));

  tbdsf_dp = gpu_calloc_int (P->numTrgBox * 2);
  tbdsr_dp = gpu_calloc_int (numAugTrg * 3);
  cs_dp = gpu_calloc_int (numSrcBoxTot);
  cp_dp = gpu_calloc_int (numSrcBoxTot);

 
  //Put data into the device
  gpu_copy_cpu2gpu_float (s_dp, P->src_, P->numSrc * (3+srcDOF));
  gpu_copy_cpu2gpu_float (t_dp, P->trg_, P->numTrg * 3);

  gpu_copy_cpu2gpu_int (tbdsf_dp, tbdsf, 2 * P->numTrgBox);
  gpu_copy_cpu2gpu_int (tbdsr_dp, tbdsr, 3 * numAugTrg);
  gpu_copy_cpu2gpu_int (cs_dp, cs, numSrcBoxTot);
  gpu_copy_cpu2gpu_int (cp_dp, cp, numSrcBoxTot);

  //kernel call
  int GRID_WIDTH=(int)ceil((float)numAugTrg/65535.0F);
  int GRID_HEIGHT=(int)ceil((float)numAugTrg/(float)GRID_WIDTH);
  dim3 BlockDim (BLOCK_HEIGHT,BLOCK_WIDTH);  //Block width will be 1
  dim3 GridDim (GRID_HEIGHT, GRID_WIDTH);    //Grid width should be 1
  //fprintf (stdout, "@@ [%s:%lu::p%d] numAugTrg=%d; BlockDim x GridDim = [%d x %d] x [%d x %d]\n", __FILE__, (unsigned long)__LINE__, mpirank, numAugTrg, BLOCK_HEIGHT, BLOCK_WIDTH, GRID_HEIGHT, GRID_WIDTH);

#if defined (__DEVICE_EMULATION__)
  GPU_MSG (">>> Device emulation mode <<<\n");
#endif
  if (numAugTrg) // No need to call kernel if numAugTrg == 0
  switch(P->kernel_type)
  {
    case KNL_LAP_S_U:
      ulist_kernel<<<GridDim,BLOCK_HEIGHT>>>(t_dp,trgVal_dp,s_dp,tbdsr_dp,tbdsf_dp,cs_dp,cp_dp,numAugTrg,1/P->kernel_coef[0]); GPU_CE;
      break;
    case KNL_STK_S_U:
      ulist_kernel_stokes_velocity<<<GridDim,BLOCK_HEIGHT>>>(t_dp,trgVal_dp,s_dp,tbdsr_dp,tbdsf_dp,cs_dp,cp_dp,numAugTrg,1/P->kernel_coef[0]); GPU_CE;
      break;
    case KNL_STK_F_U:
      ulist_kernel_stokes_fmm<<<GridDim,BLOCK_HEIGHT>>>(t_dp,trgVal_dp,s_dp,tbdsr_dp,tbdsf_dp,cs_dp,cp_dp,numAugTrg,1/P->kernel_coef[0]); GPU_CE;
      break;
    default:
      MPI_ASSERT(false);
  }

  gpu_copy_gpu2cpu_float (P->trgVal, trgVal_dp, P->numTrg*trgDOF);

  hipFree(s_dp); GPU_CE;
  hipFree(t_dp); GPU_CE;

  hipFree(trgVal_dp); GPU_CE;
  hipFree(tbdsf_dp); GPU_CE;
  hipFree(tbdsr_dp); GPU_CE;
  hipFree(cs_dp); GPU_CE;
  hipFree(cp_dp); GPU_CE;

  free(cs);
  free(cp);
  free(tbdsf);
  free(tbdsr);
}
#endif
//}//end extern
